#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <iostream>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <vector>
//#include "lattice_constans.h"



void SplitNet(p_data& P, p0_data& P0, l_data& L, pAdd_data& Padd, potential_data& Po, pNet_data& Pnet)
{
	//fprintf(stderr, "Start createLattice\n");
	P.h_F = nullptr;
	P.h_V = nullptr;
	P.h_U = nullptr;

	P.d_F = nullptr;
	P.d_V = nullptr;
	P.d_U = nullptr;


	P0.h_RU0 = nullptr;
	//P0.h_Ri = nullptr;

	P0.d_RU0 = nullptr;
	P0.d_U0 = nullptr;
	//P0.d_Ri = nullptr;


	Padd.h_Fmm = (float*)malloc(ResultFmmNum * (Padd.StepVN[0] + Padd.StepVN[1] + Padd.StepVN[2]) * sizeof(float));
	Padd.h_V = (float*)malloc((Padd.StepVN[0] + Padd.StepVN[1] + Padd.StepVN[2]) * sizeof(float));

	//Padd.d_Fmm = nullptr;
	//Padd.d_Fstx = nullptr;
	//Padd.d_Fsty = nullptr;
	//Padd.d_Fetx = nullptr;
	//Padd.d_Fety = nullptr;
	//Padd.d_cmax = nullptr;
	//Padd.d_cmin = nullptr;

	//Pnet.ScShift = int(Pnet.L/Pnet.a_aver);
#ifndef pre_OnlyOneCell
	P.N = Pnet.SN + 2;
#endif // !pre_OnlyOneCell
#ifdef pre_OnlyOneCell
	P.N = Pnet.SN + 1;
#endif // pre_OnlyOneCell


	P.h_U = (float*)malloc(2 * P.N * sizeof(float));
	P.h_V = (float*)malloc(2 * P.N * sizeof(float));
	P.h_F = (float*)malloc(2 * P.N * sizeof(float));
	P0.N = P.N;
	P0._1d_N = P._1d_N;
	P0.h_RU0 = (float*)malloc(2 * P.N * sizeof(float));

	memset(P.h_U, 0, 2 * P.N * sizeof(float));
	memset(P.h_V, 0, 2 * P.N * sizeof(float));
	memset(P.h_F, 0, 2 * P.N * sizeof(float));	
	int_fast32_t i, j, k, kk, m, mm[3], mmm;

#ifndef pre_OnlyOneCell
	double r[3][2] = { { -Pnet.CellDistance * 0.5 * ReadCoordinatesCoefficient,0 }, { Pnet.CellDistance * 0.5 * ReadCoordinatesCoefficient,0 }, { 0,0 } },//0.121052175
		rr[3] = { 0.03987475 * 0.03987475 * RCC2, 0.03987475 * 0.03987475 * RCC2, Pnet.Rcut * Pnet.Rcut * RCC2 },
		drr[3] = { 1e-4 * RCC2, 1e-4 * RCC2, 2e-2 * RCC2 },
		drx, dry, drm, _1d_rm;
	P.NBP = 2; P.NBPT[0] = 1; P.NBPT[1] = 1; P.NBPT[2] = 0;
#endif // !pre_OnlyOneCell
#ifdef pre_OnlyOneCell
	double r[3][2] = { { 0,0 }, { 1e30,1e30 }, { 0,0 } },
		rr[3] = { 0.03987475 * 0.03987475, 0.03987475 * 0.03987475, Pnet.Rcut * Pnet.Rcut },
		drr[3] = { 1e-4, 1e-4, 2e-2 },
		drx, dry, drm, _1d_rm;
	P.NBP = 1; P.NBPT[0] = 1; P.NBPT[1] = 0; P.NBPT[2] = 0;
#endif // pre_OnlyOneCell	
#ifdef pre_AlignCell
	double rn, an, nx, ny;
#endif // pre_AlignCell
	for (i = 0; i < Pnet.SN; ++i)
	{
		P0.h_RU0[i] = Pnet.h_S[i];
		P0.h_RU0[i + P.N] = Pnet.h_S[i + Pnet.SN];
		for (m = 0; m < 3; ++m)
		{
			drx = P0.h_RU0[i] - r[m][0];
			dry = P0.h_RU0[i + P.N] - r[m][1];
			drm = drx * drx + dry * dry;
			if (fabs(drm - rr[m]) < drr[m])
			{
#ifdef pre_AlignCell				
				//rn = sqrt(drm);
				if (m < 2)
				{
					//std::cerr << "R " << drx << " " << dry << " " << sqrt(drm) - 0.03987475 << " | ";
					an = -1 + sqrt(rr[m] / drm);
					P0.h_RU0[i] = r[m][0] + drx + an * drx;
					P0.h_RU0[i + P.N] = r[m][1] + dry + an * dry;
					drx = P0.h_RU0[i] - r[m][0];
					dry = P0.h_RU0[i + P.N] - r[m][1];
					drm = drx * drx + dry * dry;
					//std::cerr << drx << " " << dry << " " << sqrt(drm) - 0.03987475 << "\n";
				}				
#endif // pre_AlignCell
				++P.NBP;
				++P.NBPT[m];
			}
		}
		//std::cerr<<"P "<< P0.h_RU0[i + Pnet.ScShift]<< " "
	}
	//std::cin.get();
#ifndef pre_OnlyOneCell
	P0.h_RU0[P.N - 2] = r[0][0];
	P0.h_RU0[2 * P.N - 2] = r[0][1];
	P0.h_RU0[P.N - 1] = r[1][0];
	P0.h_RU0[2 * P.N - 1] = r[1][1];
#endif // !pre_OnlyOneCell
#ifdef pre_OnlyOneCell
	P0.h_RU0[P.N - 1] = r[0][0];
	P0.h_RU0[2 * P.N - 1] = r[0][1];
#endif // pre_OnlyOneCell

	P.h_BP = (int*)malloc(P.NBP * sizeof(int));
	P.h_BPR = (float*)malloc(2 * P.NBP * sizeof(float));
	std::cerr << "BP " << P.NBP << " " << P.NBPT[0] << " " << P.NBPT[1] << " " << P.NBPT[2] << " | " << r[0][0] << " " << r[1][0] << "\n";
	double mmax=0, mmin=1e39, minPdt=1e99, varPdt, maxC = 0, varC;
	k = 0;
	mm[0] = 0; mm[1] = 0; mm[2] = 0;
	
	/**/
#ifndef pre_OnlyOneCell
	P.h_BP[P.NBPT[0] - 1] = P.N - 2;
	P.h_BPR[P.NBPT[0] - 1] = P0.h_RU0[P.N - 2];
	P.h_BPR[P.NBPT[0] - 1 + P.NBP] = P0.h_RU0[2 * P.N - 2];
	P.h_BP[P.NBPT[0] + P.NBPT[1] - 1] = P.N - 1;
	P.h_BPR[P.NBPT[0] + P.NBPT[1] - 1] = P0.h_RU0[P.N - 1];
	P.h_BPR[P.NBPT[0] + P.NBPT[1] - 1 + P.NBP] = P0.h_RU0[2 * P.N - 1];
#endif // !pre_OnlyOneCell
#ifdef pre_OnlyOneCell
	P.h_BP[P.NBPT[0] - 1] = P.N - 1;
	P.h_BPR[P.NBPT[0] - 1] = P0.h_RU0[P.N - 1];
	P.h_BPR[P.NBPT[0] - 1 + P.NBP] = P0.h_RU0[2 * P.N - 1];
#endif // pre_OnlyOneCell
#ifdef pre_ConnectCellSurface
	//unsigned int panN = (P.NBPT[0] > P.NBPT[1]) ? P.NBPT[0] : P.NBPT[1];
	double *pangle = (double*)malloc((P.NBPT[0] + P.NBPT[1]) * sizeof(double));
#endif // pre_ConnectCellSurface
	for (i = 0; i < P.N; ++i)
	{
		mmm = 0;
		for (m = 0; m < 3; ++m)
		{
			drx = P0.h_RU0[i] - r[m][0];
			dry = P0.h_RU0[i + P.N] - r[m][1];
			drm = drx * drx + dry * dry;
			if (fabs(drm - rr[m]) < drr[m])
			{
				//if (m == 0)
				//	std::cerr << "BP " << i << " " << m << " " << mm[m] << " " << mmm << " | " << P0.h_RU0[i] << " " << P0.h_RU0[i + P.N] << "\n";
				P.h_BP[mm[m] + mmm] = i;
				P.h_BPR[mm[m] + mmm] = P0.h_RU0[i];
				P.h_BPR[mm[m] + mmm + P.NBP] = P0.h_RU0[i + P.N];
#ifdef pre_ConnectCellSurface
				if (m < 2 && drm>1e-10)
				{
					_1d_rm = 1.0 / sqrt(drm);
					if (dry * _1d_rm > 0)
						pangle[mm[m] + mmm] = acos(drx * _1d_rm);
					else
						pangle[mm[m] + mmm] = 2.0 * MC_pi - acos(drx * _1d_rm);
					//std::cerr << "BP " << m << " " << mm[m] << " | " << 180.0 * (pangle[mm[m] + mmm] * MC_1d_pi) << " | " << drx << " " << dry << " " << drm << "\n";
				}				
#endif // pre_ConnectCellSurface
				++mm[m];
				//goto L__EndCicle;
			}
			mmm += P.NBPT[m];
			//std::cin.get();
		}
	}
	/*std::cin.get();
	for (i = 0; i < P.NBPT[0] - 1; ++i)
		std::cerr << "PBR0 " << P.h_BPR[i] << " " << P.h_BPR[i + P.N] << " | " << sqrt((P.h_BPR[i] + 0.121052175) * (P.h_BPR[i] + 0.121052175) + P.h_BPR[i + P.N] * P.h_BPR[i + P.N]) << "\n";
	std::cin.get();/**/
#ifdef pre_ConnectCellSurface
#ifndef pre_OnlyOneCell
	unsigned int ACSc = P.NBPT[0] - 1 + P.NBPT[1] - 1, iACSc;
#endif // !pre_OnlyOneCell
#ifdef pre_OnlyOneCell
	unsigned int ACSc = P.NBPT[0] - 1, iACSc;
#endif // DEBUG	
	unsigned int* h_Sc_new = (unsigned int*)malloc(2 * (Pnet.ScN + ACSc) * sizeof(unsigned int));
	memcpy(h_Sc_new, Pnet.h_Sc, Pnet.ScN * sizeof(unsigned int));
	memcpy(h_Sc_new + Pnet.ScN + ACSc, Pnet.h_Sc + Pnet.ScN, Pnet.ScN * sizeof(unsigned int));
	unsigned int* h_Sc_t = Pnet.h_Sc;
	Pnet.h_Sc = h_Sc_new;
	free(h_Sc_t); h_Sc_t = nullptr; h_Sc_new = nullptr;
	P.h_BPDfi = (float*)malloc((P.NBPT[0] + P.NBPT[1]) * sizeof(float));
	memset(P.h_BPDfi, 0, (P.NBPT[0] + P.NBPT[1]) * sizeof(float));
	double dpa, dpamin=1e+30;
	unsigned int ibpn;
	mmm = 0;
	iACSc = 0;
	for (m = 0; m < 2; ++m)
	{		
		for (i = 0; i < P.NBPT[m]-1; ++i)
		{
			dpamin = 2.0 * MC_pi;
			for (j = 0; j < P.NBPT[m]-1; ++j)
			{
				if (i == j)continue;
				dpa = pangle[j + mmm] - pangle[i + mmm];
				if (dpa < 0) dpa += 2.0 * MC_pi;
				if (dpamin > dpa)
				{
					dpamin = dpa;
					ibpn = j;
				}
			}
			Pnet.h_Sc[Pnet.ScN + iACSc] = P.h_BP[i + mmm];
			Pnet.h_Sc[Pnet.ScN + iACSc + ACSc + Pnet.ScN] = P.h_BP[ibpn + mmm];
			dpa = pangle[ibpn + mmm] - pangle[i + mmm];
			if (dpa < 0) dpa += 2.0 * MC_pi;
			P.h_BPDfi[i + mmm] += 0.5 * dpa;
			P.h_BPDfi[ibpn + mmm] += 0.5 * dpa;
			++iACSc;

			//if(i + mmm> P.NBPT[0] + P.NBPT[1] || ibpn + mmm > P.NBPT[0] + P.NBPT[1])std::cerr<<"ERR!!!\n";
			//std::cerr << "BPC " << i << " " << ibpn << " | " << pangle[i + mmm] << " " << pangle[ibpn + mmm] << " | " << P.h_BP[i + mmm] << " " << P.h_BP[ibpn + mmm] 
			//	<<" | "<<sqrt((P0.h_RU0[P.h_BP[i + mmm]]- P0.h_RU0[P.h_BP[ibpn + mmm]])* (P0.h_RU0[P.h_BP[i + mmm]] - P0.h_RU0[P.h_BP[ibpn + mmm]]))<< "\n";
			//std::cin.get();
		}
		mmm += P.NBPT[m];
	}
	Pnet.ScN += ACSc;
	std::cerr << "BPSC " << iACSc << " " << P.NBPT[0] - 1 + P.NBPT[1] - 1 << "\n";
	/*dpa = 0;
	for (i = 0; i < P.NBPT[0] + P.NBPT[1]; ++i)
	{
		dpa += P.h_BPDfi[i];
		std::cerr << "Angle " << i << " " << P.h_BPDfi[i] << " " << dpa << "\n";
		if (i == P.NBPT[0])dpa = 0;
	}/**/
	free(pangle); pangle = nullptr;
#endif // pre_ConnectCellSurface
	//std::cin.get();

	P.NI = 2 * Pnet.ScN;
	P.h_In = (int*)malloc(2 * P.NI * sizeof(int));
	P.h_ShIn = (int*)malloc(2 * P.N * sizeof(int));
	P.h_Ir0 = (float*)malloc(3 * P.NI * sizeof(float));
	P.h_IM = (float*)malloc(P.N * sizeof(float));
	P.h_1d_IM = (float*)malloc(P.N * sizeof(float));
	P.h_VisR = (float*)malloc(P.N * sizeof(float));

	for (i = 0; i < P.N; ++i)
	{	
		P.h_ShIn[i] = k;		
		P.h_IM[i] = 0;
		P.h_VisR[i] = 0;
		kk = 0;
		maxC = 0;
		for (j = 0; j < Pnet.ScN; ++j)
		{
			if (Pnet.h_Sc[j] == i)
			{
				P.h_In[k] = i;// +Pnet.ScShift;
				P.h_In[k + P.NI] = Pnet.h_Sc[j + Pnet.ScN];// +Pnet.ScShift;
				drx = P0.h_RU0[P.h_In[k + P.NI]] - P0.h_RU0[i];
				dry = P0.h_RU0[P.h_In[k + P.NI] + P.N] - P0.h_RU0[i + P.N];
				P.h_Ir0[k] = drx;
				P.h_Ir0[k + P.NI] = dry;
				drm = sqrt(drx * drx + dry * dry);
				/*if (drm > 1e-1)
				{
					std::cerr << "ERR1 " << drm << " " << P.h_In[k] << " " << P.h_In[k + P.NI] << "\n";
					std::cin.get();
				}/**/
				P.h_Ir0[k + 2 * P.NI] = float(1.0 / ((1.0 - Pnet.InitialDeformation) * drm));
				P.h_IM[i] += 0.5 * Po.m * drm;
				P.h_VisR[i] += 0.5 * Po.Sfiber * drm;
				varC = Po.c / drm;
				if (varC > maxC) maxC = varC;
				//std::cerr << "A " << P.In[k] << " " << P.In[k + P.NI] << "\n";
				++k;
				++kk;
			}
			else if (Pnet.h_Sc[j + Pnet.ScN] == i)
			{
				P.h_In[k] = i;// +Pnet.ScShift;
				P.h_In[k + P.NI] = Pnet.h_Sc[j];// +Pnet.ScShift;
				drx = P0.h_RU0[P.h_In[k + P.NI]] - P0.h_RU0[i];
				dry = P0.h_RU0[P.h_In[k + P.NI] + P.N] - P0.h_RU0[i + P.N];
				P.h_Ir0[k] = drx;
				P.h_Ir0[k + P.NI] = dry;
				drm = sqrt(drx * drx + dry * dry);
				/*if (drm > 1e-1)
				{
					std::cerr << "ERR2 " << drm << " " << P.h_In[k] << " " << P.h_In[k + P.NI] << "\n";
					std::cin.get();
				}/**/
				P.h_Ir0[k + 2 * P.NI] = float(1.0 / ((1.0 - Pnet.InitialDeformation) * drm));
				P.h_IM[i] += 0.5 * Po.m * drm;
				//std::cerr << "B " << P.In[k] << " " << P.In[k + P.NI] << "\n";
				P.h_VisR[i] += 0.5 * Po.Sfiber * drm;
				varC = Po.c / drm;
				if (varC > maxC) maxC = varC;
				++k;
				++kk;
			}
		}/**/
		P.h_1d_IM[i] = 1.0f / P.h_IM[i];
		P.h_VisR[i] = pow(0.75 * P.h_VisR[i] * MC_1d_pi, MC_1d3);
		P.h_ShIn[i + P.N] = kk;
		if (mmax < P.h_IM[i]) mmax = P.h_IM[i];
		//if (mmin > P.h_IM[i]) std::cerr << "Mmin " << i << " " << P.h_IM[i] << "\n";
		if (mmin > P.h_IM[i] && P.h_IM[i]>1e-12) mmin = P.h_IM[i];
		varPdt = 0.01 * MC_pi * sqrt(P.h_IM[i] / maxC);
		//std::cerr << "maxC " << maxC << " " << varPdt<< " " << P.h_IM[i] << "\n";
		//std::cin.get();
		if (minPdt > varPdt) minPdt = varPdt;
		//std::cerr << "M " << i << " " << P.h_IM[i] << "\n";
		//std::cin.get();
//L__EndCicle:
		if(i%10000==0)std::cerr << "i "<< i <<" "<< P.NI << " " << k << " "<< P.N << "\n";
		//std::cin.get();
	}
#ifndef pre_Relaxation
	P.h_1d_IM[P.N - 2] = 0.1 / mmax;
	P.h_1d_IM[P.N - 1] = 0.1 / mmax;
#endif // !pre_Relaxation	
#ifdef pre_Relaxation
	P.h_1d_IM[P.N - 2] = 0.1 / mmax;
	P.h_1d_IM[P.N - 1] = 0.1 / mmax;
#endif // pre_Relaxation
	//std::cin.get();
	
	std::cerr << "BP LAST A " << P.h_BP[P.NBPT[0] - 1] << " " << P.h_BPR[P.NBPT[0] - 1] << " " << P.h_BPR[P.NBPT[0] - 1 + P.NBP] << "\n";
	std::cerr << "BP LAST B " << P.h_BP[P.NBPT[0] + P.NBPT[1] - 1] << " " << P.h_BPR[P.NBPT[0] + P.NBPT[1] - 1] << " " << P.h_BPR[P.NBPT[0] + P.NBPT[1] - 1 + P.NBP] << "\n";
	std::cerr << "I " << P.NI << " " << k << "\n";
	std::cerr << "Mass " << mmin << " " << mmax << " | " << Po.m << "\n";
	std::cerr << "dT " << Po.dt << " " << minPdt << " " << maxC << "\n";
	Po.dt = minPdt;
	//std::cin.get();
	P.NI = k;

	//std::cerr << "BB " << P0.h_RU0[0] << " " << P0.h_RU0[P.N] << "\n";
	std::cerr << "Data " << (6 * P.N * sizeof(float) + 2 * P.N * sizeof(int) + 5 * P.NI * sizeof(float) + 2 * P.NI * sizeof(int))/(1024*1024) << "Mb\n";
	//std::cin.get();
	//for (m = 0; m < P.NBPT[0]; ++m)	
	//	std::cerr << "BP " << m << " " << P.h_BPR[m] << " " << P.h_BPR[m + P.NBP] << "\n";		
	//std::cin.get();
	mmax = -1e39;
	for (i = 0; i < P.NBPT[0]; ++i)
	{
		if (P.h_BPR[i] > mmax)
		{
			k = i;
			mmax = P.h_BPR[i];
		}

	}
	P.iBP[0] = k;
	mmax = 1e39;
	for (i = P.NBPT[0]; i < P.NBPT[0] + P.NBPT[1]; ++i)
	{
		if (P.h_BPR[i] < mmax)
		{
			k = i;
			mmax = P.h_BPR[i];
		}
	}
	P.iBP[1] = k;
	std::cerr << "BP " << P.iBP[0] << " " << P.h_BP[P.iBP[0]] << " " << P0.h_RU0[P.h_BP[P.iBP[0]]] << "\n";
	std::cerr << "BP " << P.iBP[1] << " " << P.h_BP[P.iBP[1]] << " " << P0.h_RU0[P.h_BP[P.iBP[1]]] << "\n";
	//std::cerr << "PN " << P.N << "\n";
	//std::cin.get();
	HANDLE_ERROR(hipMalloc((void**)&P.d_U, 2 * P.N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_V, 2 * P.N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_F, 2 * P.N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_In, 2 * P.NI * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_ShIn, 2 * P.N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_Ir0, 3 * P.NI * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_1d_IM, P.N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_BP, P.NBP * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_BPR, 2 * P.NBP * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&P.d_VisR, P.N * sizeof(float)));

	std::cerr << "Fin create\n";
	//std::cin.get();
	//std::cin.get();
	HANDLE_ERROR(hipMemset((void*)P.d_V, 0, 2 * P.N * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(P.d_In, P.h_In, 2 * P.NI * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(P.d_ShIn, P.h_ShIn, 2 * P.N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(P.d_Ir0, P.h_Ir0, 3 * P.NI * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(P.d_1d_IM, P.h_1d_IM, P.N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(P.d_BP, P.h_BP, P.NBP * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(P.d_BPR, P.h_BPR, 2 * P.NBP * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(P.d_VisR, P.h_VisR, P.N * sizeof(float), hipMemcpyHostToDevice));

	//std::cerr << "FR " << Padd.d_FResult << " " << Padd.h_FResult << " " << Padd.ElementSteps * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << " " << Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << "\n";
	HANDLE_ERROR(hipMalloc((void**)&Padd.d_FResult, Padd.ElementSteps * ResultFRNum* (P.NBPT[0] + P.NBPT[1]) * sizeof(float)));
	//std::cerr<<"Data2 "<< (Padd.ElementSteps * 2 * (P.NBPT[0] + P.NBPT[1]) * sizeof(float)) / (1024.0 * 1024.0) << "Mb "<< P.NBPT[0] + P.NBPT[1] << "\n";
	Padd.h_FResult = (float*)malloc(Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) * sizeof(float));
	//std::cerr << "FR " << Padd.d_FResult << " " << Padd.h_FResult << "\n";
#ifdef pre_CalcFullEnergy
	Padd.bloks4 = P.N / (4 * SMEMDIM) + 1;
	Padd.h_Esum = (float*)malloc(2 * Padd.bloks4 * sizeof(float));
	HANDLE_ERROR(hipMalloc((void**)&Padd.d_Esum, 2 * Padd.bloks4 * sizeof(float)));
#endif // pre_CalcFullEnergy


#ifdef pre_SaveLammps
	Padd.h_Fbound0 = (float*)malloc(P.NI * 1 * sizeof(float));
	Padd.h_Fbound = (float*)malloc(P.NI * 1 * sizeof(float));
	HANDLE_ERROR(hipMalloc((void**)&Padd.d_Fbound, P.NI * 1 * sizeof(float)));
	Padd.h_LammpsAddParticles = (float*)malloc((P.NI / 2) * 4 * sizeof(float));
	Padd.h_LammpsSumF = (float*)malloc(P.N * 1 * sizeof(float));
#endif // pre_SaveLammps
#ifdef pre_SaveEnergyData
	//Padd.h_Ebound0 = (float*)malloc(P.NI * 1 * sizeof(float));
	Padd.h_Ebound = (float*)malloc(P.NI * 1 * sizeof(float));
	HANDLE_ERROR(hipMalloc((void**)&Padd.d_Ebound, P.NI * 1 * sizeof(float)));
#endif // pre_SaveEnergyData

	
	fprintf(stderr, "Finish createLattice %i\n", P.N);
	//std::cin.get();
}