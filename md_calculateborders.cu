#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>

__global__ void d_calculateBorders(const float* __restrict__ _1d_Mass, const int* __restrict__ BP, const float* __restrict__ BPR, float* __restrict__ F, float* __restrict__ V, float* __restrict__ U, float* __restrict__ FR,
	const unsigned int n, const unsigned int nbp, const unsigned int nbpt1, const unsigned int nbpt2, const unsigned int nbpt3, const unsigned int Step, const float P_vis, const float V1)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x, i, iA, iB;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//printf("B %i %i\n", idx, nbp);
	float rx, ry, _1d_rm, rrm, rr0m, m;
	if (idx < nbp)
	{
		/*if (idx == 700)
			{
				//Fetx[nt] = F[700];
				//Fety[nt] = F[700+n];

				//printf("F00 %e %e\n", F[500], F[500 + n]);
				//printf("F0 %e %e\n", F[990], F[990+n]);
				//printf("F1 %e %e\n", F[995], F[995 + n]);
				//printf("F2 %e %e\n", F[999], F[999 + n]);
			}/**/
			//printf("F %i | %e %e\n", idx, F[idx], F[idx + n]);
		i = BP[idx];
		iA = n - 2;
		iB = n - 1;

		if (idx < nbpt1 - 1)
		{

			rx = BPR[nbpt1 - 1] - BPR[idx] + U[iA] - U[i];
			ry = BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp] + U[iA + n] - U[i + n];
			//rux = rx - U[i];
			//ruy = ry - U[i + n];
			///rum = rux * rux + ruy * ruy - 0.000397498921890625f; //0.001287896506925625f;(0.9)
			//rum = rux * rux + ruy * ruy - 0.0143620218536932355625f
			//F[i] = -1.0f;
			//F[i + n] = -2.0f;			
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
#endif // !pre_FreeCell	
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
			//FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			//FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
#endif // pre_FreeCell
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;

			//printf("B %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
#ifndef pre_FreeCellHalf
			_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
			V[i] = V1 * rx * _1d_rm;
			V[i + n] = V1 * ry * _1d_rm;
			//m = __frcp_rn(_1d_Mass[i]);
			F[i] = 0.0;// 6.0f * MCf_pi * P_vis * VisR[i] * V[i] * m;
			F[i + n] = 0.0;// 6.0f * MCf_pi * P_vis * VisR[i] * V[i + n] * m;
#endif // !pre_FreeCellHalf
#ifdef pre_FreeCellHalf
			if (V1 > 1e-12)
			{
				_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
				V[i] = V1 * rx * _1d_rm;
				V[i + n] = V1 * ry * _1d_rm;
				m = 1.0f / _1d_Mass[i];
				F[i] = P_vis * V[i] * m;
				F[i + n] = P_vis * V[i + n] * m;
			}
			else
			{
				rrm = rx * rx + ry * ry;
				rr0m = (BPR[nbpt1 - 1] - BPR[idx]) * (BPR[nbpt1 - 1] - BPR[idx]) + (BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp]) * (BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp]);
				//printf("BP %i %i | %e |%e %e | %e\n", idx, i, V1, rrm, rr0m, rrm-rr0m);
				if (rrm > rr0m)
				{
					//_1d_rm = __frsqrt_rn(rrm);
					V[i] = 0;
					V[i + n] = 0;
					F[i] = 0;
					F[i + n] = 0;
				}
			}
#endif // pre_FreeCellHalf



			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, 1.0f/_1d_rm, BPR[idx], BPR[idx + nbp]);
		}
		else if (idx >= nbpt1 && idx < nbpt1 + nbpt2 - 1)
		{
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;/**/
#endif // !pre_FreeCell			
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			//if(i==4822)
			//	printf("B %i %i | %e %e %e | %e %e | %e %e\n", idx, i, F[i], F[i+n], _1d_Mass[i], V[i], V[i+n], U[i], U[i+n]);			
#endif // pre_FreeCell
			/*rx = 0.121052175f - BPR[idx];
			ry = -BPR[idx + nbp];
			FR[idx + 2 * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (2 * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			//printf("B %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
			_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
			V[i] = V1 * rx * _1d_rm;
			V[i + n] = V1 * ry * _1d_rm;
			m = 1.0f / _1d_Mass[i];
			F[i] = P_vis * V[i] * m;
			F[i + n] = P_vis * V[i + n] * m;/**/
		}/**/
		else
		{
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;
		}
		/*if (idx == nbpt1-1)
		{
			F[iA] = 0;	F[iA + n] = 0;	V[iA] = 0;	V[iA + n] = 0;
		}
		else if (idx == nbpt1 + nbpt2 - 1)
		{
			F[iB] = 0;	F[iB + n] = 0;	V[iB] = 0;	V[iB + n] = 0;
		}/**/
	}
}

__global__ void d_calculateBordersVis(const float* __restrict__ _1d_Mass, const float* __restrict__ VisR, const int* __restrict__ BP, const float* __restrict__ BPR, float* __restrict__ F, float* __restrict__ V, float* __restrict__ U, float* __restrict__ FR,
	const unsigned int n, const unsigned int nbp, const unsigned int nbpt1, const unsigned int nbpt2, const unsigned int nbpt3, const unsigned int Step, const float P_vis, const float V1)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x, i, iA, iB;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//printf("B %i %i\n", idx, nbp);
	float rx, ry, _1d_rm, rrm, rr0m, m;
	if (idx < nbp)
	{
		/*if (idx == 700)
			{
				//Fetx[nt] = F[700];
				//Fety[nt] = F[700+n];

				//printf("F00 %e %e\n", F[500], F[500 + n]);
				//printf("F0 %e %e\n", F[990], F[990+n]);
				//printf("F1 %e %e\n", F[995], F[995 + n]);
				//printf("F2 %e %e\n", F[999], F[999 + n]);
			}/**/
		//printf("F %i | %e %e\n", idx, F[idx], F[idx + n]);
		i = BP[idx];
		iA = n - 2;
		iB = n - 1;
		
		if (idx < nbpt1 - 1)
		{
			
			rx = BPR[nbpt1 - 1] - BPR[idx] + U[iA] - U[i];
			ry = BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp] + U[iA + n] - U[i + n];
			//rux = rx - U[i];
			//ruy = ry - U[i + n];
			///rum = rux * rux + ruy * ruy - 0.000397498921890625f; //0.001287896506925625f;(0.9)
			//rum = rux * rux + ruy * ruy - 0.0143620218536932355625f
			//F[i] = -1.0f;
			//F[i + n] = -2.0f;			
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
#endif // !pre_FreeCell	
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
			//FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			//FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
#endif // pre_FreeCell
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;

			//printf("B %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
#ifndef pre_FreeCellHalf
			_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
			V[i] = V1 * rx * _1d_rm;
			V[i + n] = V1 * ry * _1d_rm;
			//m = __frcp_rn(_1d_Mass[i]);
			F[i] = 6.0f * MCf_pi * P_vis * VisR[i] * V[i];
			F[i + n] = 6.0f * MCf_pi * P_vis * VisR[i] * V[i + n];
#endif // !pre_FreeCellHalf
#ifdef pre_FreeCellHalf
			if (V1 > 1e-12)
			{
				_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
				V[i] = V1 * rx * _1d_rm;
				V[i + n] = V1 * ry * _1d_rm;
				m = 1.0f / _1d_Mass[i];
				F[i] = P_vis * V[i] * m;
				F[i + n] = P_vis * V[i + n] * m;
			}
			else
			{
				rrm = rx * rx + ry * ry;
				rr0m = (BPR[nbpt1 - 1] - BPR[idx]) * (BPR[nbpt1 - 1] - BPR[idx]) + (BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp]) * (BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp]);
				//printf("BP %i %i | %e |%e %e | %e\n", idx, i, V1, rrm, rr0m, rrm-rr0m);
				if (rrm > rr0m)
				{
					//_1d_rm = __frsqrt_rn(rrm);
					V[i] = 0;
					V[i + n] = 0;
					F[i] = 0;
					F[i + n] = 0;
				}
			}
#endif // pre_FreeCellHalf

			
			
			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, 1.0f/_1d_rm, BPR[idx], BPR[idx + nbp]);
		}
		else if (idx >= nbpt1 && idx < nbpt1 + nbpt2 - 1)
		{		
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;/**/
#endif // !pre_FreeCell			
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			//if(i==4822)
			//	printf("B %i %i | %e %e %e | %e %e | %e %e\n", idx, i, F[i], F[i+n], _1d_Mass[i], V[i], V[i+n], U[i], U[i+n]);			
#endif // pre_FreeCell
			/*rx = 0.121052175f - BPR[idx];
			ry = -BPR[idx + nbp];			
			FR[idx + 2 * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (2 * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			//printf("B %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
			_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
			V[i] = V1 * rx * _1d_rm;
			V[i + n] = V1 * ry * _1d_rm;
			m = 1.0f / _1d_Mass[i];
			F[i] = P_vis * V[i] * m;
			F[i + n] = P_vis * V[i + n] * m;/**/
		}/**/
		else
		{
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;
		}		
		/*if (idx == nbpt1-1)
		{
			F[iA] = 0;	F[iA + n] = 0;	V[iA] = 0;	V[iA + n] = 0;
		}
		else if (idx == nbpt1 + nbpt2 - 1)
		{
			F[iB] = 0;	F[iB + n] = 0;	V[iB] = 0;	V[iB + n] = 0;
		}/**/
	}	
}

__global__ void d_calculateBorders(const float* __restrict__ _1d_Mass, const int* __restrict__ BP, const float* __restrict__ BPR, float* __restrict__ F, float* __restrict__ V, float* __restrict__ U, float* __restrict__ FR,
	const unsigned int n, const unsigned int nbp, const unsigned int nbpt1, const unsigned int nbpt2, const unsigned int nbpt3, const unsigned int Step, const float P_vis, const float V1, const int ibp)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x, i, iA, iB;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//printf("B %i %i\n", idx, nbp);
	float rx, ry, _1d_rm, rux, ruy, rum, m;
	if (idx < nbp)
	{		
		i = BP[idx];
		iA = n - 2;
		iB = n - 1;

		if (idx < nbpt1 - 1)
		{



			//printf("B %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
			rx = BPR[nbpt1 - 1] - BPR[idx] + U[iA] - U[i];
			ry = BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp] + U[iA + n] - U[i + n];			
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
#endif // !pre_FreeCell	
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
			//FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			//FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
#endif // pre_FreeCell
			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, 1.0f / _1d_rm, BPR[idx], BPR[idx + nbp]);
			if (idx == ibp)
			{
				//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, 1.0f / _1d_rm, BPR[idx], BPR[idx + nbp]);
				_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
				V[i] = V1 * rx * _1d_rm;
				V[i + n] = V1 * ry * _1d_rm;
				m = 1.0f / _1d_Mass[i];
				F[i] = P_vis * V[i] * m;
				F[i + n] = P_vis * V[i + n] * m;
				//F[iA] = 0;	F[iA + n] = 0;
				//F[iB] = 0;	F[iB + n] = 0;
			} else
			{
				F[i] = 0;
				F[i + n] = 0;
				V[i] = 0;
				V[i + n] = 0;
			}
			//printf("B %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);			
		}
		else if (idx >= nbpt1 && idx < nbpt1 + nbpt2 - 1)
		{
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;
#endif // !pre_FreeCell			
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			//printf("B %i %i | %e %e %e | %e %e | %e %e\n", idx, i, F[i], F[i+n], _1d_Mass[i], V[i], V[i+n], U[i], U[i+n]);			
#endif // pre_FreeCell
		}
		else
		{
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;
		}		
	}
}

__global__ void d_calculateBordersFix(const float* __restrict__ _1d_Mass, const int* __restrict__ BP, const float* __restrict__ BPR, float* __restrict__ F, float* __restrict__ V, float* __restrict__ U, float* __restrict__ FR,
	const unsigned int n, const unsigned int nbp, const unsigned int nbpt1, const unsigned int nbpt2, const unsigned int nbpt3, const unsigned int Step, const float P_vis, const float V1)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x, i, iA, iB;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//printf("B %i %i\n", idx, nbp);
	float rx, ry, _1d_rm, rrm, rr0m, m;
	if (idx < nbp)
	{
		
		i = BP[idx];
		iA = n - 2;
		iB = n - 1;

		if (idx < nbpt1 - 1)
		{

			rx = BPR[nbpt1 - 1] - BPR[idx] + U[iA] - U[i];
			ry = BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp] + U[iA + n] - U[i + n];
				
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
#endif // !pre_FreeCell	
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
#endif // pre_FreeCell
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;

			
			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, 1.0f/_1d_rm, BPR[idx], BPR[idx + nbp]);
		}
		else if (idx >= nbpt1 && idx < nbpt1 + nbpt2 - 1)
		{
#ifndef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;			
#endif // !pre_FreeCell			
#ifdef pre_FreeCell
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = U[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = U[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			//printf("B %i %i | %e %e %e | %e %e | %e %e\n", idx, i, F[i], F[i+n], _1d_Mass[i], V[i], V[i+n], U[i], U[i+n]);
#endif // pre_FreeCell			
		}		
		V[i] = 0;
		V[i + n] = 0;
		F[i] = 0;
		F[i + n] = 0;
	}
}