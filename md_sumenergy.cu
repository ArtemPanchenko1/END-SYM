#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>
//#include <cudpp.h>
//#include <cudpp_plan.h>

__device__ float d_calculatePEnergyIk(const int* __restrict__ In, const float* __restrict__ Ir0, const float* __restrict__ U, const unsigned int n, const unsigned int ni, const int ks, const int kmax, const float P_c)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k;
	float drx, dry, drm, rmada, _1d_a, c, energy;
	//unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	energy = 0;
	for (k = ks; k < kmax; ++k)
	{
		i = In[k];
		j = In[k + ni];

		drx = Ir0[k] + U[j] - U[i];
		dry = Ir0[k + ni] + U[j + n] - U[i + n];		
		_1d_a = Ir0[k + 2 * ni];
		drm = __fsqrt_rn(drx * drx + dry * dry);
		//_1d_drm = __frcp_rn(drm);

		c = P_c;
#ifdef pre_nonlinearC
		rmada = drm * _1d_a - 1.0f;
		c = dd_nonlinearC(rmada, P_c);
#endif

		rmada = drm - __frcp_rn(_1d_a);
		//fsumx = c * drx * (_1d_a - _1d_drm);
		//fsumy = c * dry * (_1d_a - _1d_drm);
		//Fbound[k] = __fsqrt_rn(fsumx * fsumx + fsumy * fsumy);
		energy += c * _1d_a * rmada * rmada;
		//if(fsumx* fsumx+ fsumy* fsumy>1e-5)
		//if(idx==4818)
		//	printf("F %i %i %i | %e %e %e | %e %e | %e %e\n", idx, i, j, (_1d_a - _1d_drm),(_1d_a - _1d_drm), _1d_a, U[i], U[i+n], U[j], U[j + n]);
		//printf("FF %e %e %e \n", drx, (1.0 - P_a * _1d_drm), fx[1]);
		//printf("F %u %u | %e %e %e | %e %e | %e %e\n",  j1, idx, fx[1], fy[1], drm, U[j1], U[idx], U[j1 + n], U[idx + n]);

	}
	return energy;
}

__device__ float d_calculateKEnergyi(const float* __restrict__ V, const unsigned int i, const unsigned int n, const float m)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	//int i, j, k;
	//float drx, dry, drm, rmada, _1d_a, c, energy;
	//unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	float energy;
	energy = 0.5 * m * (V[i] * V[i] + V[i + n] * V[i + n]);
	return energy;
}

__global__ void d_getEnergyEntire(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ _1d_Mass, const float* __restrict__ U, const float* __restrict__ V, float* Esum, const unsigned int n, const unsigned int ni, const float P_c)
{
	// static shared memory
	__shared__ float s_mem[2 * SMEMDIM];

	// set thread ID
	// global index, 4 blocks of input data processed at a time
	unsigned int tid = threadIdx.x, idx = blockIdx.x * blockDim.x * 4 + threadIdx.x, i;
	int ks, kmax;
	// unrolling 4 blocks
	float ek = 0, ep = 0, m;

	// boundary check
	if (idx + 3 * blockDim.x < n)
	{
		float t_ek0 = 0, t_ek1 = 0, t_ek2 = 0, t_ek3 = 0;
		float t_ep0 = 0, t_ep1 = 0, t_ep2 = 0, t_ep3 = 0;

		i = idx;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		i = idx + blockDim.x;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		i = idx + 2 * blockDim.x;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		i = idx + 3 * blockDim.x;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy	
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);		
		ek = t_ek0 + t_ek1 + t_ek2 + t_ek3;
		ep = t_ep0 + t_ep1 + t_ep2 + t_ep3;
	}
	else if (idx + 2 * blockDim.x < n)
	{
		float t_ek0 = 0, t_ek1 = 0, t_ek2 = 0;
		float t_ep0 = 0, t_ep1 = 0, t_ep2 = 0;

		i = idx;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		i = idx + blockDim.x;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		i = idx + 2 * blockDim.x;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);		
		ek = t_ek0 + t_ek1 + t_ek2;
		ep = t_ep0 + t_ep1 + t_ep2;
	}
	else if (idx + blockDim.x < n)
	{
		float t_ek0 = 0, t_ek1 = 0;
		float t_ep0 = 0, t_ep1 = 0;

		i = idx;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		i = idx + blockDim.x;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);		
		ek = t_ek0 + t_ek1;
		ep = t_ep0 + t_ep1;
	}
	else if (idx < n)
	{
		float t_ek0 = 0;
		float t_ep0 = 0;

		i = idx;
		ks = ShIn[i];
		kmax = ks + ShIn[i + n];
		t_ep0 = d_calculatePEnergyIk(In, Ir0, U, n, ni, ks, kmax, P_c);
#ifdef pre_CalcFullKEnergy
		m = __frcp_rn(_1d_Mass[i]);
		t_ek0 = d_calculateKEnergyi(V, i, n, m);
#endif // pre_CalcFullKEnergy
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);		
		ek = t_ek0;
		ep = t_ep0;
	}/**/

	//if(idx + 5 * blockDim.x >4619700)
	//   printf("TT %i %i %i %f %i\n", tid, idx, blockIdx.x, n);
	//if (ns>1e-3f)
	//   printf("TT %i %i %f\n", tid, idx, ns);
	s_mem[tid] = ep;
	s_mem[tid + SMEMDIM] = ek;
	__syncthreads();

	//if(idx==0)
	//	printf("TT %i %f %f %i %i\n", tid, s_ek, e_ek, s_n, e_n);

	// in-place reduction in shared memory
	if (blockDim.x >= 1024 && tid < 512)
	{
		s_mem[tid] += s_mem[tid + 512];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 512];
	}
	__syncthreads();

	if (blockDim.x >= 512 && tid < 256)
	{
		//printf("Blok!\n");
		s_mem[tid] += s_mem[tid + 256];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 256];
	}
	__syncthreads();

	if (blockDim.x >= 256 && tid < 128)
	{
		s_mem[tid] += s_mem[tid + 128];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 128];
	}
	__syncthreads();

	if (blockDim.x >= 128 && tid < 64)
	{
		s_mem[tid] += s_mem[tid + 64];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 64];
	}

	__syncthreads();
	/*if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < SMEMDIM; ++i)
			printf("GM %i %e\n", i, smem[i + 3 * SMEMDIM]);
	}/**/

	// unrolling warp
	if (tid < 32)
	{
		volatile float* vsmem = s_mem;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 2];
		vsmem[tid] += vsmem[tid + 1];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 1];
	}

	// write result for this block to global mem
	if (tid == 0)
	{
		Esum[blockIdx.x] = s_mem[0];
		Esum[blockIdx.x + gridDim.x] = s_mem[SMEMDIM];
		//if (smem[tid + 3 * SMEMDIM] > 1e-3f)
		//	printf("TT %i %i %f\n", tid, idx, smem[tid + 3 * SMEMDIM]);
		//if (smem[3 * SMEMDIM] > 1e-3f)
		//printf("T %i %f\n", blockIdx.x, gridDim.x, smem[3 * SMEMDIM]);
	}/**/
}