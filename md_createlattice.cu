#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <iostream>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
//#include "lattice_constans.h"



void createLattice(p_data& P, p0_data& P0, l_data& L, pAdd_data &Padd, potential_data &Po)
{   
    /*
    //fprintf(stderr, "Start createLattice\n");
    P.h_F = nullptr;
    P.h_V = nullptr;
    P.h_U = nullptr;
    
    P.d_F = nullptr;
    P.d_V = nullptr;
    P.d_U = nullptr;

        
    P0.h_RU0 = nullptr;  
    P0.h_Ri = nullptr;

    P0.d_RU0 = nullptr;
    P0.d_U0 = nullptr;    
    P0.d_Ri = nullptr;


    Padd.h_Fmm = nullptr;
    Padd.h_V = nullptr;

    Padd.d_Fmm = nullptr;
    Padd.d_Fstx = nullptr;
    Padd.d_Fsty = nullptr;
    Padd.d_Fetx = nullptr;
    Padd.d_Fety = nullptr;
    Padd.d_cmax = nullptr;
    Padd.d_cmin = nullptr;

    //Padd.h_S = nullptr;         
    //Padd.h_Sc = nullptr;

    L.LV.x = Po.a;
    L.LV.y = 0;
    
    L.n.x = P.N;
    L.n.y = 1;

    L.PS.x = L.LV.x * float(L.n.x);
    L.PS.y = L.LV.y * float(L.n.y);
            
    fprintf(stderr, "Sample size L/a %f %f | N %i\n", L.PS.x / Po.a, L.PS.y / Po.a, P.N);
    P0.N = P.N;
    P0._1d_N = P._1d_N;

    Padd.bloks = ceil(P.N / (SMEMDIM)) + 1;

    /*int memmax = (8 * Padd.bloks4 * sizeof(float) + (11 + NGPUEk * 2) * 2 * P.N * sizeof(float)) / (1024 * 1024);
    if (memmax > 7500) 
    {
        fprintf(stderr, "Error Max Memory! %i\n", memmax); return;
    }/**/
    //P._1d_N = 1e-10;

    /*
    fprintf(stderr, "Start create Arrays %i %e | %u\n", P0.N, P._1d_N, Padd.bloks);
    P.h_U = (float*)malloc(2 * P.N * sizeof(float));
    P.h_V = (float*)malloc(2 * P.N * sizeof(float));
    P.h_F = (float*)malloc(2 * P.N * sizeof(float));     

    P0.h_Ri = (int*)malloc(2 * P.N * sizeof(int));
    P0.h_RU0 = (float*)malloc(2 * P0.N * sizeof(float));

    Padd.h_Fmm = (float*)malloc(8 * (Padd.StepV1 + Padd.StepV2) * sizeof(float));
    Padd.h_V = (float*)malloc( (Padd.StepV1 + Padd.StepV2) * sizeof(float));


#ifdef  pre_XVfb	
    Padd.MaxTimeStep = (2.0 * Padd.MaxShift / Padd.V01 + 1.1 * Po.a * (1.0 + Padd.Eps0) * P.N / Padd.Vl) / Po.dt + 1;
    Padd.MaxTimeStep = SMEMDIM * (Padd.MaxTimeStep / SMEMDIM + 1);
#endif //  pre_XVfb

#ifdef pre_YVfb
    Padd.MaxTimeStep = (2.0 * Padd.MaxShift / Padd.V01 + 1.1 * Po.a * (1.0 + Padd.Eps0) * P.N / Padd.Vt) / Po.dt + 1;
    Padd.MaxTimeStep = SMEMDIM * (Padd.MaxTimeStep / SMEMDIM + 1);    
#endif
    Padd.blokst = Padd.MaxTimeStep / (SMEMDIM);
    unsigned int memmax = (8 * (Padd.StepV1 + Padd.StepV2) * sizeof(float) + 2 * Padd.MaxTimeStep * sizeof(float) + 2 * Padd.blokst * sizeof(float)) / (1024 * 1024);
    fprintf(stderr, "Max memory GPU %u Mb | %u %e %e\n", memmax, Padd.MaxTimeStep, 1.1 * Po.a * (1.0 + Padd.Eps0) * P.N / Padd.Vt, Padd.Eps0);
    //std::cin.get();
    HANDLE_ERROR(hipMalloc((void**)&Padd.d_Fmm, 8 * (Padd.StepV1 + Padd.StepV2) * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&Padd.d_Fstx, Padd.MaxTimeStep * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&Padd.d_Fsty, Padd.MaxTimeStep * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&Padd.d_Fetx, Padd.MaxTimeStep * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&Padd.d_Fety, Padd.MaxTimeStep * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&Padd.d_cmax, Padd.blokst * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&Padd.d_cmin, Padd.blokst * sizeof(float)));
    Padd.h_Fstx = (float*)malloc(Padd.MaxTimeStep * sizeof(float));
    Padd.h_Fsty = (float*)malloc(Padd.MaxTimeStep * sizeof(float));
    Padd.h_Fetx = (float*)malloc(Padd.MaxTimeStep * sizeof(float));
    Padd.h_Fety = (float*)malloc(Padd.MaxTimeStep * sizeof(float));
    //P.N = 1000;
       
    fprintf(stderr, "Finish create Arrays\n");

    //fprintf(stderr, "Start create cuda Arrays\n");
    
   
    HANDLE_ERROR(hipMalloc((void**)&P.d_F, 2 * P.N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&P.d_U, 2 * P.N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&P.d_V, 2 * P.N * sizeof(float)));    

    HANDLE_ERROR(hipMalloc((void**)&P0.d_Ri, 2 * P0.N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&P0.d_RU0, 2 * P0.N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&P0.d_U0, 2 * P0.N * sizeof(float)));

    //fprintf(stderr, "Start createLattice_kernel\n");
    //d_createLattice <<<Padd.bloks, SMEMDIM >>> (P0.d_RU0, P0.d_Ri, P0.N, L.LV.x);
    d_createLattice << <Padd.bloks, SMEMDIM >> > (P0.d_RU0, P0.d_U0, P0.d_Ri, P0.N, L.LV.x, Padd.Eps0);

    //fprintf(stderr, "Finish createLattice_kernel\n");/**/

    //fprintf(stderr, "Start memset\n");
    //HANDLE_ERROR(hipMemset((void*)P.d_U, 0, 2 * P.N * sizeof(float)));
    //fprintf(stderr, "Finish memset\n");
/*
    //fprintf(stderr, "Start memcpy\n");
    HANDLE_ERROR(hipMemset((void*)P.d_V, 0, 2 * P.N * sizeof(float)));
    HANDLE_ERROR(hipMemcpy(P0.h_RU0, P0.d_RU0, 2 * P0.N * sizeof(float), hipMemcpyDeviceToHost));
    //fprintf(stderr, "Finish memcpy\n");

    //for (int i = 0; i < P0.N; ++i)    
    //    fprintf(stderr, "P %i %f %f | %f %f\n", i, P0.RU0X[i], P0.RU0Y[i], P0.RV0X[i], P0.RV0Y[i]);
    /*

    L.iCenter = L.n.x / 2;
    L.rCenter.x = P0.h_RU0[L.iCenter];
    L.V = P0.N * L.LV.x;
    L._1d_V = 1.0 / L.V;
    
    ///hiprandCreateGenerator(&Padd.gen, HIPRAND_RNG_PSEUDO_MTGP32);
    ///hiprandSetPseudoRandomGeneratorSeed(Padd.gen, time(NULL));

    fprintf(stderr, "Finish createLattice %i\n", P.N);  /**/  
}

__global__ void d_createLattice(float* __restrict__ RU0, float* __restrict__ U0, int* __restrict__ Ri, const int n, const float L0X, const float Eps0)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x, ix, iy;
    //printf("GPU0 %i %i | %i %f \n", i, N, nX, _1d_nX);
    float fi;// , fix, fiy;
    while (i < n)
    {
        fi = __int2float_rn(i);
          
        RU0[i] =     fi*L0X;
        RU0[i + n] = 0;

        U0[i] = fi * Eps0 * L0X;
        U0[i + n] = 0;

        Ri[i] = __float2int_rn(fi);
        Ri[i + n] = 0;
        //printf("GPU0 %i %i | %f %f %f %f | \%i %i\n", i, N, RV0X[i], RV0Y[i], RU0X[i], RU0Y[i], RiX[i], RiY[i]);
        i += blockDim.x * gridDim.x;
    }  
    return;
}

void renewLattice(p_data& P, p0_data& P0)
{         
    //HANDLE_ERROR(hipMemset((void*)P.d_U, 0, 2 * P.N * sizeof(float)));
    HANDLE_ERROR(hipMemset((void*)P.d_V, 0, 2 * P.N * sizeof(float)));
    HANDLE_ERROR(hipMemset((void*)P.d_U, 0, 2 * P.N * sizeof(float)));
    //HANDLE_ERROR(hipMemcpy((void*)P.d_U, (void*)P0.d_U0, 2 * P.N * sizeof(float), hipMemcpyDeviceToDevice));   
    //fprintf(stderr, "Finish renewLattice\n");
}