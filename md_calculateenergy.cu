#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>


__global__ void d_calculateKineticEnergy(	
	const float* __restrict__ VU, const float* __restrict__ VV,
	float* __restrict__ aEk, unsigned int offset,
	unsigned int n)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < n)
	{		
		aEk[offset +         idx] += VU[idx] *     VU[idx];
		aEk[offset +     n + idx] += VU[idx + n] * VU[idx + n];
		aEk[offset + 2 * n + idx] += VV[idx] *     VV[idx];
		aEk[offset + 3 * n + idx] += VV[idx + n] * VV[idx + n];
	}	
}

__global__ void d_calculateKineticEnergy_precision(
	const float* __restrict__ VU, const float* __restrict__ VV,
	const float* __restrict__ FU, const float* __restrict__ FV,
	float* __restrict__ aEk, unsigned int offset,
	unsigned int n, const float P_dtm)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < n)
	{
		float uvx = VU[idx] + 0.5 * FU[idx] * P_dtm;
		float uvy = VU[idx + n] + 0.5 * FU[idx + n] * P_dtm;
		float vvx = VV[idx] + 0.5 * FV[idx] * P_dtm;
		float vvy = VV[idx + n] + 0.5 * FV[idx + n] * P_dtm;
		aEk[offset + idx] += uvx * uvx;
		aEk[offset + n + idx] += uvy * uvy;
		aEk[offset + 2 * n + idx] += vvx * vvx;
		aEk[offset + 3 * n + idx] += vvy * vvy;
	}
}

