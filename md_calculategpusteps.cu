#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>
#include <chrono>
#include "m_func.h"

//#include <cudpp.h>
//#include <cudpp_plan.h>


void calculateGPUSteps(p_data& P, p0_data& P0, potential_data& Po, pAdd_data& Padd, l_data& L, pNet_data &Pnet)
{
	//std::cerr << "W1\n";
	/*hipEvent_t start, stop;
	float gpuTime;
	double gpuTimeAver = 0;/**/
	//float dV, maxShift = 10.0, Vl = sqrt(Po.a * Po.a * Po.c * Po._1d_m), Vd = 1.01;
	//Padd.V = 0.013;
	char filename[256] = "";
	size_t time, bloks, estep, esteps = ResultFRSave, Estep, esize = Padd.ElementSteps * ResultFRNum * (P.NBPT[0] + P.NBPT[1]), i, j,
		EFsize = 2 * Padd.ElementSteps * Padd.EF.NR;// , timestart, t1, t2;
	//std::cerr << "AAA! " << EFsize << " " << Padd.ElementSteps << " " << Padd.EF.NR << " " << 2 * Padd.ElementSteps << "\n";
	double d = 0;
	float v = Padd.V, fa_max, fb_min;
	bool contraction = true;
#ifdef pre_SineImpuls
	double omega = MC_pi * Padd.V / Padd.RMove, t1 = Padd.RMove / (Padd.V * Po.dt);
#endif // pre_SineImpuls

	time = Padd.time;
	//timestart = 1.01*(2.0 * Padd.MaxShift / Padd.V) / Po.dt + 1;
	//time = 10;
	//float v, v2;
	//time = 1500000;
	
	Padd.bloks = P.N / (SMEMDIM) + 2;
	Padd.bloksb = P.NBP / (SMEMDIM) + 2;
	std::cerr << "Bloks " << Padd.bloks << " " << Padd.bloksb << "\n";
	//Padd.blokst = time / (SMEMDIM);
	//t1 = Padd.MaxShift / (Padd.V * Po.dt);
	//v2 = (double(Padd.MaxShift) - t1 * Padd.V * Po.dt) / Po.dt;
	//t2 = (fabs(v2) < 1e-8) ? 0 : 1;
	//printf("cGS %u %u | %e %e %e %e | %e %e\n", time, Padd.blokst, Padd.MaxShift, 1.1 * Po.a * (1.0 + Padd.Eps0) * P.N / Padd.Vl, Padd.Vl, Po.dt, Padd.V, Po.vism);

	std::cerr << "0 Step" << " " << v << " " << d << " " << time << " | " << unsigned int(0.016235485 / Po.dt) << " " << Padd.ImpulsSteps + unsigned int(0.016235485 / Po.dt) << " " << "\n";
#ifdef pre_SaveLammps
	d_calculateForcesIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Fbound, P.N, P.NI, Po.c);
	hipMemcpy(Padd.h_Fbound0, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
	d_calculateUIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ubound, P.N, P.NI, Po.c);
	hipMemcpy(Padd.h_Ubound0, Padd.d_Ubound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
	unsigned int stepssavelamms = unsigned int(0.005 * 0.016235485 / Po.dt);
	stepssavelamms = unsigned int(Padd.time / 500);
	stepssavelamms = 5000;
	
	std::cerr << "SSLammps " << stepssavelamms << " " << Padd.ImpulsSteps + stepssavelamms * 100 << " | " << (Padd.ImpulsSteps + stepssavelamms / 0.005) / stepssavelamms << "\n"; //std::cin.get();
#ifdef pre_SaveLammpsEnergy
	d_calculateEnergyIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ebound, P.N, P.NI, Po.c);
	hipMemcpy(Padd.h_Ebound0, Padd.d_Ebound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
	for (i = 0; i < P.N; ++i)
		Padd.h_Ek0[i] = P.h_IM[i] * (P.h_V[i] * P.h_V[i] + P.h_V[i + P.N] * P.h_V[i + P.N]);
#endif // pre_SaveLammpsEnergy	
#endif // pre_SaveLammps
#ifdef pre_SaveEnergyData
	//std::cerr << "A1\n"; std::cin.get();
	unsigned int stepssaveenergy = unsigned int(0.01 * 0.016235485 / Po.dt);
	stepssaveenergy = 18144;//0.001
	stepssaveenergy = 22889;//0.005
	stepssaveenergy = 30316;//0.01
	//stepssaveenergy = 23959;//0.05
	//stepssaveenergy = 46282;//0.1
	//stepssaveenergy = 45266;//0.5
	
	std::cerr << "SSEnergy " << stepssaveenergy <<" "<< stepssaveenergy* Po.dt << "\n";
	stepssaveenergy *= 0.01;
	//std::cin.get();
	d_calculateEnergyIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ebound, P.N, P.NI, Po.c);
	//std::cerr << "A2\n"; std::cin.get();
	sprintf(filename, "./result/CP_Energy.txt");
	SaveEnergyDataStart(P, P0, Pnet, Padd, Po, filename, 0);
	//std::cerr << "A3\n"; std::cin.get();
#endif // pre_SaveEnergyData
#ifdef pre_OneCellEFdistribution
	d_calculateForcesI_EF << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, Padd.EF.d_EFb0, P.N, P.NI, Po.c);
#endif // pre_OneCellEFdistribution
	
	//std::cerr << "T1 " << time * Po.dt << " " << t1 << " " << Po.dt << "\n";
	Estep = 0;
	//esteps = 0;
	Po.CShfreefiber = 1.0 / (double(0.5 * Po.hfreefiber) * log(0.5 * double(Po.hfreefiber) / double(Po.rfiber)));
#ifdef pre_CylinderViscocityShapovalov
	std::cerr << "Po.CShfreefiber " << Po.CShfreefiber << " " << Po.hfreefiber << " " << Po.rfiber << "\n";
#endif // pre_CylinderViscocityShapovalov
	
	for (unsigned long int steps = 0; steps < time; ++steps)
	{
		if (steps % 100000 == 0)std::cerr << "Steps " << steps << "\n";
		//if (steps % 10 == 0)++esteps;
		estep = (steps/ esteps) % Padd.ElementSteps;
		if (estep >= Padd.ElementSteps)std::cerr << "AAAA! " << estep << " " << esize << "\n";
		//d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_U, P.d_F, P.N, Po.c, Po.a, Po._1d_a);
		//d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (const int* __restrict__ In, const float* __restrict__ Ir0, const float* __restrict__ U, float* __restrict__ IF, const unsigned int n, const unsigned int ni, const float P_c, const float P_a, const float P_1d_a);
		//d_sumForcesI << < Padd.bloks, SMEMDIM >> > (const int* __restrict__ In, const float* __restrict__ IF, float* F, const unsigned int n, const unsigned int ni)
		//std::cerr << "Q1\n";
#ifndef pre_OneCellEFdistribution
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
#ifdef pre_CylinderViscocity
#ifdef pre_CylinderViscocity1
		d_calculateVIscosForces << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_V, P.d_F, P.N, P.NI, Po.vis, 1.0 / Po.vis, float(Po.rfiber), float(Po.roliquid));
#endif // pre_CylinderViscocity1
#ifdef pre_CylinderViscocity2
		d_calculateVIscosForces2 << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_V, P.d_F, P.N, P.NI, Po.vis, 1.0 / Po.vis, float(Po.rfiber), float(Po.roliquid), float(2.0 / Po.hfreefiber));
#endif // pre_CylinderViscocity2
#ifdef pre_CylinderViscocity3
		d_calculateVIscosForces3 << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_V, P.d_F, P.N, P.NI, Po.vis, 1.0 / Po.vis, float(Po.rfiber), float(Po.roliquid), float(2.0 / Po.hfreefiber));
#endif // pre_CylinderViscocity3
#ifdef pre_CylinderViscocityShapovalov
		d_calculateVIscosForcesShapovalov << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_V, P.d_F, P.N, P.NI, Po.vis, 1.0 / Po.vis, float(Po.rfiber), float(Po.roliquid), Po.CShfreefiber);
#endif // pre_CylinderViscocityShapovalov
#ifdef pre_CylinderViscocityLindstrom
		d_calculateVIscosForcesLindstrom << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_V, P.d_F, P.N, P.NI, Po.vis, float(Po.rfiber), float(1.0 / Po.rfiber), float(Po.roliquid));
		//std::cin.get();
#endif // pre_CylinderViscocityLindstrom
#endif // pre_CylinderViscocity

		
		

#endif // !pre_OneCellEFdistribution
		
#ifdef pre_OneCellEFdistribution
		//std::cerr << "start distr "<<estep<<"\n";
		d_calculateForcesI_EF << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, Padd.EF.d_EFb, P.N, P.NI, Po.c);
		d_distrubuteBoundToCircle << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_U, P0.d_RU0, Padd.EF.d_EFb, Padd.EF.d_EFb0, Padd.EF.d_CEF, P.N, P.NI, estep, Padd.EF.P00[0], Padd.EF.P00[1], Padd.EF._1d_DR);
		//std::cerr << "E " << estep << " " << 2 * NumberCircles * estep << "\n";
		//std::cin.get();
#endif // pre_OneCellEFdistribution


		//std::cerr << "Q2\n";/
#ifndef pre_MoveCell
#ifndef pre_OneNodeContract
		d_calculateBordersVis << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_VisR, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, 0.0f, v);//Po.vis=0
#endif // !pre_OneNodeContract
#ifdef pre_OneNodeContract
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v, P.iBP[0]);
#endif // pre_OneNodeContract

#endif // !pre_MoveCell
#ifdef pre_MoveCell
#ifndef pre_OneNodeContract
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // !pre_OneNodeContract
#ifdef pre_OneNodeContract
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v, P.iBP[0]);
#endif // pre_OneNodeContract
#endif // pre_MoveCell	
		//std::cerr << "Q3\n";
		d_calculateIncrementsVis << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_VisR, P.d_F, P.d_V, P.d_U, P.N, Po.dt, 0.0f);//Po.vis=0
		//d_calculateIncrementsMove << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, Padd.d_FResult, P.d_V, P.d_U, P.N, Po.dt, Po.vis, estep);
		//std::cerr << "Q4\n";
		//std::cin.get();
		//if (steps % 10000 == 0)	std::cerr << "steps " << steps << "\n";
		/*if (steps % 1000 == 0)
		{
			std::cerr << steps << " " << v << " | " << d << " " << Padd.RMove << "\n";
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			//hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);
		}/**/
#ifdef pre_SaveLammps
#ifdef pre_SaveLammpsPoint
		//if (steps % stepssavelamms == 0 && abs(int(steps) - int(Padd.LammpsPointSaveTime)) - 10 < stepssavelamms)
		if (steps == Padd.LammpsPointSaveTime)
#endif // pre_SaveLammpsPoint
#ifndef pre_SaveLammpsPoint
		if (steps % stepssavelamms == 0)
#endif // !pre_SaveLammpsPoint		
		{
			std::cerr << "Fin Step" << steps << " " << v << " " << d << "\n";
			d_calculateForcesIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Fbound, P.N, P.NI, Po.c);
			//std::cin.get();

			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
			//std::cin.get();
#ifdef pre_SaveLammpsEnergy
			d_calculateEnergyIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ebound, P.N, P.NI, Po.c);
			//std::cin.get();
			hipMemcpy(Padd.h_Ebound, Padd.d_Ebound, P.NI * sizeof(float), hipMemcpyDeviceToHost);

#endif // pre_SaveLammpsEnergy			
#ifdef pre_SaveLammpsU
			d_calculateUIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ubound, P.N, P.NI, Po.c);
			//std::cin.get();
			hipMemcpy(Padd.h_Ubound, Padd.d_Ubound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
#endif // pre_SaveLammpsU	
			//sprintf(filename, "./result/steps/CP_%li.txt", steps);
			//SaveTXTParticles(P, P0, Po, Pnet, filename);
			sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
			SaveLammpsDATAParticles(P, P0, Po, Pnet, Padd, filename);
			//std::cin.get();
		}
#endif // pre_SaveLammps	
#ifdef pre_SaveEnergyData
		if (steps % stepssaveenergy == 0)
		{
			std::cerr << "Fin Step" << steps << " " << v << " " << d << "\n";
			d_calculateEnergyIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ebound, P.N, P.NI, Po.c);

			//hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
			//sprintf(filename, "./result/steps/CP_%li.txt", steps);
			//SaveTXTParticles(P, P0, Po, Pnet, filename);
			sprintf(filename, "./result/CP_Energy.txt");
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, Padd, filename);
			SaveEnergyDataStep(P, P0, Pnet, Padd, Po, filename, steps);
			//std::cin.get();
		}
#endif // pre_SaveEnergyData
		//std::cerr << "Q5\n";
		if (estep + 1 == Padd.ElementSteps && steps > 0 && steps% esteps ==0)
		//if (estep + 1 == Padd.ElementSteps && steps > 0)
		{
			//std::cerr << "AAA! "<< steps <<" "<< Estep << " " << esize << "\n";
			hipMemcpy(Padd.h_FResult + Estep * esize, Padd.d_FResult, esize * sizeof(float), hipMemcpyDeviceToHost);
#ifdef pre_OneCellEFdistribution
			//if(esteps%10==0)
			hipMemcpy(Padd.EF.h_CEF + Estep * EFsize, Padd.EF.d_CEF, EFsize * sizeof(float), hipMemcpyDeviceToHost);
			HANDLE_ERROR(hipMemset((void*)Padd.EF.d_CEF, 0, Padd.ElementSteps * 2 * Padd.EF.NR * sizeof(float)));
#endif // pre_OneCellEFdistribution			
			++Estep;
		}/**/
		//std::cerr << "Q6\n";
		/**if (steps % StepsToGPU == 0)
		{
			ai = steps / StepsToGPU;
			//fprintf(stderr, "calculateGPUSteps %i %i %i\n", steps, steps % StepsToGPU, ai);
			d_calculateKineticEnergy <<< Padd.bloks, SMEMDIM >>> (P.d_VU, P.d_VV, Padd.d_Ek, (4 * P.N * ai), P.N);
			//d_calculateKineticEnergy_precision <<< Padd.bloks, SMEMDIM >>> (P.d_VU, P.d_VV, P.d_FU, P.d_FV, Padd.d_Ek, (4 * P.N * ai), P.N, Po.dtm);
		}/**/
		//if (steps % 100 == 0)std::cin.get();
		//std::cin.get();
		/*if (fabs(steps - t1) < 1 || steps==0)
		{
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			double rrr = sqrt((P0.h_RU0[P.h_BP[10]] + P.h_U[P.h_BP[10]] + 0.121052175) * (P0.h_RU0[P.h_BP[10]] + P.h_U[P.h_BP[10]] + 0.121052175) + (P0.h_RU0[P.h_BP[10] + P0.N] + P.h_U[P.h_BP[10] + P.N]) * (P0.h_RU0[P.h_BP[10] + P0.N] + P.h_U[P.h_BP[10] + P.N]));
			std::cerr << "Check " << steps << " " << steps * Po.dt << " " << t1 << " " << Po.dt << " | " << P.h_BP[10] << " " << P0.h_RU0[P.h_BP[10]] << " " << P0.h_RU0[P.h_BP[10] + P0.N]
				<< " " << P.h_U[P.h_BP[10]] << " " << P.h_U[P.h_BP[10] + P.N]
				<< " | " << rrr << " " << rrr+Padd.RMove << " " << rrr - Padd.RMove - 0.03987475 <<" " << Padd.RMove << "\n";
			std::cin.get();
		}/**/
#ifdef pre_SineImpuls
		
#ifndef pre_ReleaseHalfSine
		if (steps < 2 * t1) v = 0.5 * Padd.RMove * omega * sin(omega * Po.dt * steps);
		else v = 0;/**/
#endif // !pre_ReleaseHalfSine
#ifdef pre_ReleaseHalfSine
		if (steps < t1) v = -0.5 * Padd.RMove * omega * sin(omega * Po.dt * steps);
		else v = 0;/**/
#endif // pre_ReleaseHalfSine
		//v = 0;
#endif // pre_SineImpuls
#ifdef pre_SquareImpuls
		if (contraction)
		{
			d += v * Po.dt;
			if (d + v * Po.dt > Padd.RMove)
			{
				v = (Padd.RMove - d) / Po.dt;
				//std::cerr << "VV " << v << " " << d << "\n";
				///d = Padd.RMove;
				contraction = false;
			}
		}
		else
		{
#ifdef TwoDirectionMove
			d += v * Po.dt;
			if (v > 0)
			{
				v = -Padd.V;
			}
			else if (d < 1e-12)
			{
				v = 0;
				d = 0;
			}
			else if (d + v * Po.dt < 0)
			{
				v = -d / Po.dt;
			}
#endif // TwoDirectionMove
#ifndef TwoDirectionMove
			v = 0;
#endif // !TwoDirectionMove		
		}		
#endif // pre_SquareImpuls
		//if(estep%10000==0)
		//if(v<1e-12)
		//std::cin.get();
		//if (steps % 10 == 0)std::cerr << steps << "\n";
	}
	//std::cerr << "AA! "<< Estep * esize << " " << Estep << " " << esize << " " << (estep + 1) * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << " " << ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << "\n";
	hipMemcpy(Padd.h_FResult + Estep * esize, Padd.d_FResult, (estep + 1)* esize* size_t(sizeof(float)), hipMemcpyDeviceToHost);
	//std::cerr << "AAA! " << Estep << " " << EFsize << " " << Padd.ElementSteps << " " << Padd.EF.NR << " " << estep << "\n";
#ifdef pre_OneCellEFdistribution
	hipMemcpy(Padd.EF.h_CEF + Estep * EFsize, Padd.EF.d_CEF, (estep + 1) * 2 * Padd.EF.NR * sizeof(float), hipMemcpyDeviceToHost);
	EFMinMaxData(P, P0, Pnet, Padd, Po);
#endif // pre_OneCellEFdistribution		
	/*for (j = 0; j < Padd.time; j += 500)
	{
		std::cerr << "RE";
		for (i = 0; i < Padd.EF.NR; ++i)
		{
			std::cerr << " " << Padd.EF.h_CEF[j * 2 * Padd.EF.NR + i];
		}
		std::cerr << "\n";
		std::cin.get();
	}/**/
	//std::cin.get();


	calculate_Fminmax2(Padd, P, P0, Po);

	//hipMemcpy(Padd.h_Fstx, Padd.d_Fstx, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fsty, Padd.d_Fsty, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fetx, Padd.d_Fety, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fety, Padd.d_Fety, time * sizeof(float), hipMemcpyDeviceToHost);
#ifdef pre_SaveFR
	SaveTXTGraphsFR(Padd, P, P0, Po,10);
#endif // pre_SaveFR



	//std::cin.get();/**/

	//HANDLE_ERROR(hipMemcpy(Padd.h_Ek, Padd.d_Ek, (NGPUEk * 4 * P0.N * sizeof(float)), hipMemcpyDeviceToHost));
	//HANDLE_ERROR(hipMemset((void*)Padd.d_Ek, 0, (NGPUEk * 4 * P0.N * sizeof(float))));	
}

void calculateGPUStepsContractRelax(p_data& P, p0_data& P0, potential_data& Po, pAdd_data& Padd, l_data& L, pNet_data& Pnet)
{
	//std::cerr << "W1\n";
	/*hipEvent_t start, stop;
	float gpuTime;
	double gpuTimeAver = 0;/**/
	//float dV, maxShift = 10.0, Vl = sqrt(Po.a * Po.a * Po.c * Po._1d_m), Vd = 1.01;
	///Padd.V = 2e-2;
	char filename[256] = "";
	unsigned int time, bloks, estep, Estep, esize = Padd.ElementSteps * ResultFRNum * (P.NBPT[0] + P.NBPT[1]), i, j, steps, stepStop = Padd.RelaxationTime;// , timestart, t1, t2;
	double d = 0, povis0 = Po.vis;
	float v = Padd.V, fa_max, fb_min, Ep0 = 2.21821;
	bool contraction = true;


	time = Padd.time;
	//timestart = 1.01*(2.0 * Padd.MaxShift / Padd.V) / Po.dt + 1;

	//float v, v2;
	//time = 1500000;

	Padd.bloks = P.N / (SMEMDIM)+2;
	Padd.bloksb = P.NBP / (SMEMDIM)+2;

	std::cerr << "Bloks " << Padd.bloks << " " << Padd.bloksb << "\n";
	//Padd.blokst = time / (SMEMDIM);
	//t1 = Padd.MaxShift / (Padd.V * Po.dt);
	//v2 = (double(Padd.MaxShift) - t1 * Padd.V * Po.dt) / Po.dt;
	//t2 = (fabs(v2) < 1e-8) ? 0 : 1;
	//printf("cGS %u %u | %e %e %e %e | %e %e\n", time, Padd.blokst, Padd.MaxShift, 1.1 * Po.a * (1.0 + Padd.Eps0) * P.N / Padd.Vl, Padd.Vl, Po.dt, Padd.V, Po.vism);
	std::chrono::steady_clock::time_point begin = std::chrono::high_resolution_clock::now();
	std::chrono::steady_clock::time_point end;
	std::chrono::nanoseconds dr;
	Estep = 0;
#ifdef pre_SaveLammps
	d_calculateForcesIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Fbound, P.N, P.NI, Po.c);
	hipMemcpy(Padd.h_Fbound0, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
#endif // pre_SaveLammps
	
#ifdef pre_SaveEnergyDataRelax
	//std::cerr << "A1\n"; std::cin.get();
	d_calculateEnergyIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ebound, P.N, P.NI, Po.c);
	//std::cerr << "A2\n"; std::cin.get();
	sprintf(filename, "./result/CP_Energy.txt");
	SaveEnergyDataStart(P, P0, Pnet, Padd, Po, filename, 0);
	//std::cerr << "A3\n"; std::cin.get();
#endif // pre_SaveEnergyData
#ifdef pre_CalcFullEnergy
	sprintf(filename, "./result/CP_Energy.txt");
	std::ofstream file_energy_txt;
	file_energy_txt.open(filename, std::ios::out);
	file_energy_txt << "step time Ep Ek Efull\n";
	file_energy_txt.precision(10);
#endif // pre_CalcFullEnergy

	

	
#ifdef pre_OneNodeContractRelax
	double rMoveOne = 1.0 * 0.01 * 0.03987475 * ReadCoordinatesCoefficient, v1;
	unsigned int onecontracttime = unsigned int(rMoveOne / (v * Po.dt));
	v1 = rMoveOne / (onecontracttime * Po.dt);
	std::cerr << "Time_OCC " << onecontracttime << " " << rMoveOne << " " << v1 << "\n";
	for (steps = 0; steps < onecontracttime; ++steps)
	{		
		estep = 0;
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
		//std::cerr << "Q2\n";
#ifdef pre_OneNodeContract
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v1, P.iBP[0]);
#endif // pre_OneNodeContract
		d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);

		//if (steps % 100000 == 0)std::cerr << "steps " << steps << "\n";
		/*if (steps % 10000 == 0)
		{
			end = std::chrono::high_resolution_clock::now();
			dr = end - begin;
			std::cerr << "Fin Step" << steps << " " << v << " " << d << " | " << std::chrono::duration_cast<std::chrono::milliseconds>(dr).count() <<" ms " << "\n";
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);
			begin = std::chrono::high_resolution_clock::now();
			std::cin.get();
			//sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, filename);
		}/**/

	}
#ifdef pre_SaveLammps
	//if (steps % 100000 == 0)
	{
		std::cerr << "Fin Step" << steps << " " << v << " " << d << "\n";
		d_calculateForcesIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Fbound, P.N, P.NI, Po.c);


		hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
		//sprintf(filename, "./result/steps/CP_%li.txt", steps);
		//SaveTXTParticles(P, P0, Po, Pnet, filename);
		sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
		SaveLammpsDATAParticles(P, P0, Po, Pnet, Padd, filename);
	}
#endif // pre_SaveLammps
#endif // pre_OneNodeContractRelax

	//std::cerr << "AAAAA!!!\n";
	//std::cin.get();
	if (d + v * Po.dt > Padd.RMove)
	{
		v = 0;
		contraction = false;
	}
	//for (unsigned int steps = 0; steps < 4000001; ++steps)
	for (steps = 0; steps < Padd.RelaxationTime; ++steps)
	{
		estep = 0;
		if (contraction)
		{			
			if (d + v * Po.dt > Padd.RMove)
			{
				v = (Padd.RMove - d) / Po.dt;
				//std::cerr << "VV " << v << " " << d << "\n";
				///d = Padd.RMove;
				contraction = false;
				stepStop = steps;
			}
			d += v * Po.dt;
		}
		else					
			v = 0;
		
		//d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_U, P.d_F, P.N, Po.c, Po.a, Po._1d_a);
		//d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (const int* __restrict__ In, const float* __restrict__ Ir0, const float* __restrict__ U, float* __restrict__ IF, const unsigned int n, const unsigned int ni, const float P_c, const float P_a, const float P_1d_a);
		//d_sumForcesI << < Padd.bloks, SMEMDIM >> > (const int* __restrict__ In, const float* __restrict__ IF, float* F, const unsigned int n, const unsigned int ni)
		//std::cerr << "Q1\n";
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
		//std::cerr << "Q2\n";
#ifndef pre_MoveCell
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // !pre_MoveCell
#ifdef pre_MoveCell
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // pre_MoveCell	
		//std::cerr << "Q3\n";
		d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);
		//std::cerr << "Q4\n";
		//std::cin.get();
#ifdef pre_CalcFullEnergy
		if ((steps - stepStop) % 1000 == 0 && steps >= stepStop)
		{
			d_getEnergyEntire << < Padd.bloks4, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_1d_IM, P.d_U, P.d_V, Padd.d_Esum, P.N, P.NI, Po.c);
			hipMemcpy(Padd.h_Esum, Padd.d_Esum, 2 * Padd.bloks4 * sizeof(float), hipMemcpyDeviceToHost);
			Padd.Esum[0] = 0;
			Padd.Esum[1] = 0;
			for (i = 0; i < Padd.bloks4; ++i)
			{
				Padd.Esum[0] += Padd.h_Esum[i];
				Padd.Esum[1] += Padd.h_Esum[i + Padd.bloks4];
			}
			
			file_energy_txt << steps - stepStop << " " << (steps - stepStop) * Po.dt << " " << Padd.Esum[0] - Ep0 << " " << Padd.Esum[1] << " " << Padd.Esum[0] - Ep0 + Padd.Esum[1] << "\n";
			if (steps % 10000 == 0)
				std::cerr << "E " << Padd.Esum[0] << " " << Padd.Esum[1] << " | " << contraction << " " << v << "\n";
			//if (steps == stepStop)Ep0 = Padd.Esum[0];
			
			//if (steps % 2000000 == 0)
			//	std::cin.get();
		}/**/
#endif // pre_CalcFullEnergy

		/*if (steps == 500000)
		{
			Po.vis = 0;
			Po.vism = Po.vis * Po.m;
		}
		else if (steps == 1000000)
		{
			Po.vis = povis0;
			Po.vism = Po.vis * Po.m;
		}
		else if (steps > 1000000 && steps % 50000 == 0)
		{
			Po.vis *= 1.01;
			Po.vism = Po.vis * Po.m;
		}/**/
		if (steps > 1000000 && steps % 200000 == 0)
		{
			hipMemset(P.d_V, 0, 2 * P.N * sizeof(float));			
		}/**/
		/*if (steps == 4000000)
		{
			Po.vis = 0;
			Po.vism = Po.vis * Po.m;
		}/**/
		if (steps > 499999 && steps % 10000 == 0)
		{
			Po.vis = povis0 * (0.5 - 0.5 * sin((steps - 500000) * 1e-5 * MC_pi + 1.5 * MC_pi));
			//std::cerr << "VIs " << Po.vis << " " << ((steps - 500000) * 1e-5 * MC_pi + 1.5 * MC_pi) / MC_pi << "\n";
			Po.vism = Po.vis * Po.m;
			//std::cin.get();
		}
		//if (steps % 100000 == 0)std::cerr << "steps " << steps << "\n";
		/*if (steps % 10000 == 0)
		{
			end = std::chrono::high_resolution_clock::now();
			dr = end - begin;			
			std::cerr << "Fin Step" << steps << " " << v << " " << d << " | " << std::chrono::duration_cast<std::chrono::milliseconds>(dr).count() <<" ms " << "\n";
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);
			begin = std::chrono::high_resolution_clock::now();
			std::cin.get();
			//sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, filename);
		}/**/
#ifdef pre_SaveLammps
		if (steps % 100000 == 0)
		{
			std::cerr << "Fin Step" << steps << " " << v << " " << d << "\n";
			d_calculateForcesIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Fbound, P.N, P.NI, Po.c);


			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
			//sprintf(filename, "./result/steps/CP_%li.txt", steps);
			//SaveTXTParticles(P, P0, Po, Pnet, filename);
			sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, Padd, filename);
		}
#endif // pre_SaveLammps

#ifdef pre_SaveEnergyDataRelax
		if (steps % 10000 == 0)
		{
			std::cerr << "Fin Step" << steps << " " << v << " " << d << "\n";
			d_calculateEnergyIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ebound, P.N, P.NI, Po.c);
			

			//hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
			//sprintf(filename, "./result/steps/CP_%li.txt", steps);
			//SaveTXTParticles(P, P0, Po, Pnet, filename);
			sprintf(filename, "./result/CP_Energy.txt");
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, Padd, filename);
			SaveEnergyDataStep(P, P0, Pnet, Padd, Po, filename, steps);
			std::cin.get();
		}		
#endif // pre_SaveEnergyData
		//std::cerr << "Q5\n";
		/*if (estep == 0 && steps > 0)
		{
			hipMemcpy(Padd.h_FResult + Estep * esize, Padd.d_FResult, esize * sizeof(float), hipMemcpyDeviceToHost);
			std::cerr << "AAA!\n";
			++Estep;
		}/**/
		//std::cerr << "Q6\n";
		/**if (steps % StepsToGPU == 0)
		{
			ai = steps / StepsToGPU;
			//fprintf(stderr, "calculateGPUSteps %i %i %i\n", steps, steps % StepsToGPU, ai);
			d_calculateKineticEnergy <<< Padd.bloks, SMEMDIM >>> (P.d_VU, P.d_VV, Padd.d_Ek, (4 * P.N * ai), P.N);
			//d_calculateKineticEnergy_precision <<< Padd.bloks, SMEMDIM >>> (P.d_VU, P.d_VV, P.d_FU, P.d_FV, Padd.d_Ek, (4 * P.N * ai), P.N, Po.dtm);
		}/**/
		//if (steps % 1000 == 0)std::cin.get();
		//std::cin.get();
		
		//std::cin.get();
	}
	Po.vis = 0;
	Po.vism = Po.vis * Po.m;
	hipMemset(P.d_V, 0, 2 * P.N * sizeof(float));
	v = 0;
	for (unsigned int steps = 0; steps < Padd.time; ++steps)
	{
		
		estep = steps % Padd.ElementSteps;		
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
#ifndef pre_MoveCell
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // !pre_MoveCell
#ifdef pre_MoveCell
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // pre_MoveCell	
		d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);	
		/*if (estep == 0 && steps > 0)
		{
			hipMemcpy(Padd.h_FResult + Estep * esize, Padd.d_FResult, esize * sizeof(float), hipMemcpyDeviceToHost);
			std::cerr << "AAA!\n";
			++Estep;
		}/**/
		//std::cin.get();
	}
	
	//std::cerr << "AA! " << Padd.h_FResult << " " << Padd.d_FResult << " " << Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << "\n";
	hipMemcpy(Padd.h_FResult, Padd.d_FResult, Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) * sizeof(float), hipMemcpyDeviceToHost);
	//std::cerr << "AA! " << Padd.h_FResult << " " << Padd.d_FResult << " " << Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << "\n";
	//std::cerr << "AA! " << Padd.h_FResult[0] << " " << Padd.h_FResult[100] << " " << Padd.h_FResult[1000] << "\n";
	//std::cerr << "AAA!\n";
	calculate_Faver2(Padd, P, P0, Po);
	//calculate_Fminmax2(Padd, P, P0, Po);
	hipMemset(P.d_V, 0, 2 * P.N * sizeof(float));

#ifdef pre_CalcFullEnergy	
	file_energy_txt.close();
#endif // pre_CalcFullEnergy
	//hipMemcpy(Padd.h_Fstx, Padd.d_Fstx, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fsty, Padd.d_Fsty, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fetx, Padd.d_Fety, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fety, Padd.d_Fety, time * sizeof(float), hipMemcpyDeviceToHost);
#ifdef pre_SaveFR
	SaveTXTGraphsFR(Padd, P, P0, Po);
#endif // pre_SaveFR

	

	//std::cin.get();/**/

	//HANDLE_ERROR(hipMemcpy(Padd.h_Ek, Padd.d_Ek, (NGPUEk * 4 * P0.N * sizeof(float)), hipMemcpyDeviceToHost));
	//HANDLE_ERROR(hipMemset((void*)Padd.d_Ek, 0, (NGPUEk * 4 * P0.N * sizeof(float))));	
}

void calculateGPUStepsAverage(p_data& P, p0_data& P0, potential_data& Po, pAdd_data& Padd, l_data& L, pNet_data& Pnet)
{	
	char filename[256] = "";
	unsigned int time, bloks, estep, Estep, esize = Padd.ElementSteps * 2 * (P.NBPT[0] + P.NBPT[1]), i, j;// , timestart, t1, t2;
	double d = 0;
	float v = Padd.V, fa_max, fb_min;
	bool contraction = true;


	time = Padd.time;
	//timestart = 1.01*(2.0 * Padd.MaxShift / Padd.V) / Po.dt + 1;

	//float v, v2;
	//time = 1500000;

	Padd.bloks = P.N / (SMEMDIM)+2;
	Padd.bloksb = P.NBP / (SMEMDIM)+2;
	std::cerr << "Bloks " << Padd.bloks << " " << Padd.bloksb << "\n";
	//Padd.blokst = time / (SMEMDIM);
	//t1 = Padd.MaxShift / (Padd.V * Po.dt);
	//v2 = (double(Padd.MaxShift) - t1 * Padd.V * Po.dt) / Po.dt;
	//t2 = (fabs(v2) < 1e-8) ? 0 : 1;
	//printf("cGS %u %u | %e %e %e %e | %e %e\n", time, Padd.blokst, Padd.MaxShift, 1.1 * Po.a * (1.0 + Padd.Eps0) * P.N / Padd.Vl, Padd.Vl, Po.dt, Padd.V, Po.vism);
	Estep = 0;
	v = 0;
#ifdef pre_OneCellEFdistribution	
	HANDLE_ERROR(hipMemset((void*)Padd.EF.d_CEF, 0, Padd.ElementSteps * 2 * Padd.EF.NR * sizeof(float)));
#endif // pre_OneCellEFdistribution
	//std::cin.get();

	/*double maxE;
	int nimax;
	d_calculateEnergyIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Ebound, P.N, P.NI, Po.c);
	hipMemcpy(Padd.h_Ebound0, Padd.d_Ebound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
	maxE = Padd.h_Ebound0[0]; nimax = 0;
	for (i = 1; i < P.NI; ++i)
	{
		//std::cerr << i << " " << Padd.h_Ebound0[i] << "\n";
		if (fabs(maxE) < fabs(Padd.h_Ebound0[i]))
		{
			maxE = Padd.h_Ebound0[i];
			nimax = i;
		}
	}
	std::cerr.precision(10);
	std::cerr << "MaxE " << maxE << " " << nimax << "\n"; std::cin.get();/**/


	for (unsigned int steps = 0; steps < Padd.time; ++steps)
	{

		estep = steps % Padd.ElementSteps;
#ifndef pre_OneCellEFdistribution
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
#endif // !pre_OneCellEFdistribution

#ifdef pre_OneCellEFdistribution
		//std::cerr << "start distr "<<estep<<"\n";
		d_calculateForcesI_EF << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, Padd.EF.d_EFb, P.N, P.NI, Po.c);
		d_distrubuteBoundToCircle << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_U, P0.d_RU0, Padd.EF.d_EFb, Padd.EF.d_EFb0, Padd.EF.d_CEF, P.N, P.NI, estep, Padd.EF.P00[0], Padd.EF.P00[1], Padd.EF._1d_DR);
		//std::cerr << "E " << estep << " " << 2 * NumberCircles * estep << "\n";
		//std::cin.get();
#endif // pre_OneCellEFdistribution
#ifndef pre_MoveCell
		//d_calculateBordersFix << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#ifndef pre_OneNodeContract
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // !pre_OneNodeContract
#ifdef pre_OneNodeContract
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v, P.iBP[0]);
#endif // pre_OneNodeContract
#endif // !pre_MoveCell
#ifdef pre_MoveCell
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // pre_MoveCell			
		d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);

		/*if (steps % 10000 == 0)
		{			
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);			
			//sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, filename);
		}/**/
		/*if (estep == 0 && steps > 0)
		{
			hipMemcpy(Padd.h_FResult + Estep * esize, Padd.d_FResult, esize * sizeof(float), hipMemcpyDeviceToHost);
			std::cerr << "AAA!\n";
			++Estep;
		}/**/
#ifdef pre_SaveLammps
		if (steps % 40000 == 0)
		{
			std::cerr << "Fin Step" << steps << " " << v << " " << d << "\n";
			d_calculateForcesIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Fbound, P.N, P.NI, Po.c);


			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
			//sprintf(filename, "./result/steps/CP_%li.txt", steps);
			//SaveTXTParticles(P, P0, Po, Pnet, filename);
			sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, Padd, filename);
		}/**/
#endif // pre_SaveLammps
		//if(steps%10000==0)
		//	d_PrintFR << < Padd.bloksb, SMEMDIM >> > (Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep);
		//std::cin.get();
	}
#ifdef pre_FreeCellHalf
	unsigned int ii;
	hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	for (i = 0; i < P.NBPT[0]-1; ++i)
	{
		ii = P.h_BP[i];
		//std::cerr << "PBR " << P.h_BPR[i] << " " << P.h_BPR[i + P.NBP] << " | " << sqrt((P.h_BPR[i] + 0.121052175) * (P.h_BPR[i] + 0.121052175) + P.h_BPR[i + P.NBP] * P.h_BPR[i + P.NBP]) << " | ";
		P.h_BPR[i] = P0.h_RU0[ii] + P.h_U[ii];
		P.h_BPR[i + P.NBP] = P0.h_RU0[ii + P.N] + P.h_U[ii + P.N];
		//std::cerr << P.h_BPR[i] << " " << P.h_BPR[i + P.N] << " | " << P.h_BPR[i] + 0.121052175 << " | " << sqrt((P.h_BPR[i] + 0.121052175) * (P.h_BPR[i] + 0.121052175) + P.h_BPR[i + P.NBP] * P.h_BPR[i + P.NBP]) << "\n";
	}
	hipMemcpy(P.d_BPR, P.h_BPR, 2 * P.NBP * sizeof(float), hipMemcpyHostToDevice);
#endif // pre_FreeCellHalf

	
	//std::cerr << "FR " << Padd.d_FResult << " " << P.N << " " << P.NBP << " " << estep <<" | "<< Padd.bloksb << " "<< SMEMDIM << "\n";
	//std::cin.get();
	
	//for (unsigned int steps = 0; steps < 10000; ++steps)
	//	d_PrintFR << < Padd.bloksb, SMEMDIM >> > (Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep);
	
	//std::cin.get();
	//std::cerr << "AA!\n";
	//std::cerr << "FR " << Padd.time <<" "<< ResultFRNum <<" "<< (P.NBPT[0] + P.NBPT[1]) <<" "<< Padd.h_FResult << "\n";
	hipMemcpy(Padd.h_FResult, Padd.d_FResult, Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) * sizeof(float), hipMemcpyDeviceToHost);
#ifdef pre_OneCellEFdistribution
	//if(esteps%10==0)
	hipMemcpy(Padd.EF.h_CEF, Padd.EF.d_CEF, Padd.time * 2 * Padd.EF.NR * sizeof(float), hipMemcpyDeviceToHost);
	HANDLE_ERROR(hipMemset((void*)Padd.EF.d_CEF, 0, Padd.ElementSteps * 2 * Padd.EF.NR * sizeof(float)));
	EFAverageData(P, P0, Pnet, Padd, Po);
#endif // pre_OneCellEFdistribution	
	hipMemset(P.d_V, 0, 2 * P.N * sizeof(float));
	//std::cerr << "AAA!\n";
	

	//std::cerr << "FR " << Padd.h_FResult[0] << " " << Padd.h_FResult[1] << " " << Padd.h_FResult[2] << "\n";
	calculate_Faver2(Padd, P, P0, Po);
	//SaveTXTGraphsFR(Padd, P, P0, Po);
	//std::cin.get();
}