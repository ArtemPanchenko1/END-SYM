#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>
//#include <cudpp.h>
//#include <cudpp_plan.h>

__global__ void d_getSinVelocityEntire(
	const float* __restrict__ i_r,
	const float* __restrict__ i_v,
	float* __restrict__ o_D,	
	unsigned int n)
{
	// static shared memory
	__shared__ float s_mem[4 * SMEMDIM];

	// set thread ID
	unsigned int tid = threadIdx.x;

	// global index, 4 blocks of input data processed at a time
	unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

	// unrolling 4 blocks
	float vx = 0, vy = 0, ek = 0, _n = 0;

	// boundary check
	if (idx + 3 * blockDim.x < n)
	{
		float vx0 = 0, vy0 = 0;
		float vx1 = 0, vy1 = 0;
		float vx2 = 0, vy2 = 0;
		float vx3 = 0, vy3 = 0;		
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);

		vx0 = i_v[idx];					 vy0 = i_v[idx + n];				 
		vx1 = i_v[idx + blockDim.x];	 vy1 = i_v[idx + blockDim.x + n];	 
		vx2 = i_v[idx + 2 * blockDim.x]; vy2 = i_v[idx + 2 * blockDim.x + n];
		vx3 = i_v[idx + 3 * blockDim.x]; vy3 = i_v[idx + 3 * blockDim.x + n];
				
		vx = vx0 + vx1 + vx2 + vx3;
		vy = vy0 + vy1 + vy2 + vy3;
		ek = vx0 * vx0 + vx1 * vx1 + vx2 * vx2 + vx3 * vx3 +
			 vy0 * vy0 + vy1 * vy1 + vy2 * vy2 + vy3 * vy3;
		_n = 4;
		
	}
	else if (idx + 2 * blockDim.x < n)
	{
		float vx0 = 0, vy0 = 0;
		float vx1 = 0, vy1 = 0;
		float vx2 = 0, vy2 = 0;
		
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);
		vx0 = i_v[idx];					 vy0 = i_v[idx + n];				 
		vx1 = i_v[idx + blockDim.x];	 vy1 = i_v[idx + blockDim.x + n];	  
		vx2 = i_v[idx + 2 * blockDim.x]; vy2 = i_v[idx + 2 * blockDim.x + n]; 
		
		vx = vx0 + vx1 + vx2;
		vy = vy0 + vy1 + vy2;
		ek = vx0 * vx0 + vx1 * vx1 + vx2 * vx2 +
			vy0 * vy0 + vy1 * vy1 + vy2 * vy2;
		_n = 3;		
	}
	else if (idx + blockDim.x < n)
	{
		float vx0 = 0, vy0 = 0, n0 = 0;
		float vx1 = 0, vy1 = 0, n1 = 0;		
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);
		vx0 = i_v[idx];					 vy0 = i_v[idx + n];			 
		vx1 = i_v[idx + blockDim.x];	 vy1 = i_v[idx + blockDim.x + n];
				
		vx = vx0 + vx1;
		vy = vy0 + vy1;
		ek = vx0 * vx0 + vx1 * vx1 +
			vy0 * vy0 + vy1 * vy1;
		_n = 2;		
	}
	else if (idx < n)
	{
		float vx0 = 0, vy0 = 0;		
	
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);
		vx0 = i_v[idx];					 vy0 = i_v[idx + n];	

		vx = vx0;
		vy = vy0;
		ek = vx0 * vx0 + vy0 * vy0;
		_n = 1;		
	}/**/
	//if(idx + 5 * blockDim.x >4619700)
	//   printf("TT %i %i %i %f %i\n", tid, idx, blockIdx.x, n);
	//if (ns>1e-3f)
	//   printf("TT %i %i %f\n", tid, idx, ns);
	s_mem[tid              ] = vx;
	s_mem[tid +     SMEMDIM] = vy;
	s_mem[tid + 2 * SMEMDIM] = ek;
	s_mem[tid + 3 * SMEMDIM] = _n;
	__syncthreads();

	//if(idx==0)
	//	printf("TT %i %f %f %i %i\n", tid, s_ek, e_ek, s_n, e_n);

	// in-place reduction in shared memory
	if (blockDim.x >= 1024 && tid < 512)
	{
		s_mem[tid              ] += s_mem[tid +               512];
		s_mem[tid +     SMEMDIM] += s_mem[tid +     SMEMDIM + 512];
		s_mem[tid + 2 * SMEMDIM] += s_mem[tid + 2 * SMEMDIM + 512];
		s_mem[tid + 3 * SMEMDIM] += s_mem[tid + 3 * SMEMDIM + 512];
	}
	__syncthreads();

	if (blockDim.x >= 512 && tid < 256)
	{
		//printf("Blok!\n");
		s_mem[tid] += s_mem[tid + 256];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 256];
		s_mem[tid + 2 * SMEMDIM] += s_mem[tid + 2 * SMEMDIM + 256];
		s_mem[tid + 3 * SMEMDIM] += s_mem[tid + 3 * SMEMDIM + 256];
	}
	__syncthreads();

	if (blockDim.x >= 256 && tid < 128)
	{

		s_mem[tid] += s_mem[tid + 128];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 128];
		s_mem[tid + 2 * SMEMDIM] += s_mem[tid + 2 * SMEMDIM + 128];
		s_mem[tid + 3 * SMEMDIM] += s_mem[tid + 3 * SMEMDIM + 128];
	}
	__syncthreads();

	if (blockDim.x >= 128 && tid < 64)
	{
		s_mem[tid] += s_mem[tid + 64];
		s_mem[tid + SMEMDIM] += s_mem[tid + SMEMDIM + 64];
		s_mem[tid + 2 * SMEMDIM] += s_mem[tid + 2 * SMEMDIM + 64];
		s_mem[tid + 3 * SMEMDIM] += s_mem[tid + 3 * SMEMDIM + 64];
	}

	__syncthreads();
	/*if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < SMEMDIM; ++i)
			printf("GM %i %e\n", i, smem[i + 3 * SMEMDIM]);
	}/**/

	// unrolling warp
	if (tid < 32)
	{
		volatile float* vsmem = s_mem;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 32];
		vsmem[tid + 2 * SMEMDIM] += vsmem[tid + 2 * SMEMDIM + 32];
		vsmem[tid + 3 * SMEMDIM] += vsmem[tid + 3 * SMEMDIM + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 16];
		vsmem[tid + 2 * SMEMDIM] += vsmem[tid + 2 * SMEMDIM + 16];
		vsmem[tid + 3 * SMEMDIM] += vsmem[tid + 3 * SMEMDIM + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 8];
		vsmem[tid + 2 * SMEMDIM] += vsmem[tid + 2 * SMEMDIM + 8];
		vsmem[tid + 3 * SMEMDIM] += vsmem[tid + 3 * SMEMDIM + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 4];
		vsmem[tid + 2 * SMEMDIM] += vsmem[tid + 2 * SMEMDIM + 4];
		vsmem[tid + 3 * SMEMDIM] += vsmem[tid + 3 * SMEMDIM + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 2];
		vsmem[tid + 2 * SMEMDIM] += vsmem[tid + 2 * SMEMDIM + 2];
		vsmem[tid + 3 * SMEMDIM] += vsmem[tid + 3 * SMEMDIM + 2];
		vsmem[tid] += vsmem[tid + 1];
		vsmem[tid + SMEMDIM] += vsmem[tid + SMEMDIM + 1];
		vsmem[tid + 2 * SMEMDIM] += vsmem[tid + 2 * SMEMDIM + 1];
		vsmem[tid + 3 * SMEMDIM] += vsmem[tid + 3 * SMEMDIM + 1];
	}

	// write result for this block to global mem
	if (tid == 0)
	{
		o_D[blockIdx.x] = s_mem[0];
		o_D[blockIdx.x + gridDim.x] = s_mem[SMEMDIM];
		o_D[blockIdx.x + 2 * gridDim.x] = s_mem[2 * SMEMDIM];
		o_D[blockIdx.x + 3 * gridDim.x] = s_mem[3 * SMEMDIM];
		//if (smem[tid + 3 * SMEMDIM] > 1e-3f)
		//	printf("TT %i %i %f\n", tid, idx, smem[tid + 3 * SMEMDIM]);
		//if (smem[3 * SMEMDIM] > 1e-3f)
		//printf("T %i %f\n", blockIdx.x, gridDim.x, smem[3 * SMEMDIM]);
	}/**/
}

__global__ void d_setSinVelocityEntire(
	const float* __restrict__ i_r, 
	const float* __restrict__ i_v,
	float* __restrict__ o_v,
	unsigned int n, float x0,  
	float A_vx, float A_vy, float vc, float vt, float vr)

	
{
	// static shared memory
	//__shared__ float smem[4 * SMEMDIM];

	// set thread ID
	unsigned int tid = threadIdx.x;

	// global index, 4 blocks of input data processed at a time
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// unrolling 4 blocks
	float dx, vcc;

	// boundary check
	if (idx < n)
	{
		//printf("T %i %i %i\n", tid, idx, blockIdx.x);
		//float a1 = g_idata[idx];		
		//printf("T3 %i %i %i\n", tid, idx, blockIdx.x);
		dx = i_r[idx] - x0;
		//dy = i_r[idx + n] - yc;		
		//r = sqrtf(dx * dx + dy * dy);
		vcc = vc * sqrtf(1.0f + vt * sinf(vr * dx));			
		o_v[idx] = vcc * (i_v[idx] - A_vx);
		o_v[idx + n] = vcc * (i_v[idx + n] - A_vy);		
	}
}


//void template <256> __global__ reduceCompleteUnroll(float* g_idata, float* g_odata, unsigned int n);

void setSinTemperatureNormal(p_data& P, p0_data& P0, param_data& Pr, potential_data& Po, pAdd_data& Padd, l_data &L)
{
	hipEvent_t start, stop;
	float gpuTime;
	double gpuTimeAver = 0;
	double average_vx=0, average_vy=0, average_ek=0, vcoeff=0, tcoeff, rcoeff;
	double n=0, x0;

	
	//thrust::device_ptr<float> d_VUXptr = thrust::device_pointer_cast(P.d_VUX);
	//hiprandCreateGenerator(&gen, HIPRAND_RNG_QUASI_SOBOL32);	
	//hiprandGenerateNormal(Padd.gen, Padd.d_SR_V, 2 * P.N, 0.0f, sqrtf(1.0f * Po._1d_m * Pr.EkSpot));
	hiprandGenerateNormal(Padd.gen, Padd.d_ER_V, 2 * P.N, 0.0f, sqrtf(1.0f * Po._1d_m * Pr.Ek));
	
	//std::cerr << "Pr " << Pr.xCenter << " " << Pr.yCenter << " " << Pr.rSpot << "\n";

	d_getSinVelocityEntire << <Padd.bloks4, SMEMDIM >> > (P0.d_RU0, Padd.d_ER_V, Padd.d_bD4, P.N);	
	
	HANDLE_ERROR(hipMemcpy(Padd.h_bD4, Padd.d_bD4, 4 * Padd.bloks4 * sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < Padd.bloks4; ++i)
	{
		average_vx += Padd.h_bD4[i];
		average_vy += Padd.h_bD4[i + Padd.bloks4];
		average_ek += Padd.h_bD4[i + 2 * Padd.bloks4];
		n += Padd.h_bD4[i + 3 * Padd.bloks4];		
		//fprintf(stderr, "n %i %i %e\n", i, bloks, vsh[4][i + bloks]);
	}
	average_vx /= n;
	average_vy /= n;
	average_ek *= 0.5 * Po.m / n;
	average_ek -= 0.5f * Po.m * (average_vx * average_vx + average_vy * average_vy);
	vcoeff = sqrt(Pr.Ek / average_ek);
	tcoeff = Pr.EkSpot / Pr.Ek;
	rcoeff = 2.0 * MC_pi / L.PS[0].x;
	x0 = P0.h_RU0[0];
	//fprintf(stderr, "N %e %e %i | %e %e %e %e | %e %e | %e %e\n", _n, environment_n, P.N, 
	//	average_vx, average_vy, environmentaverage_vx, environmentaverage_vy, 
	//	average_ek, Pr.Ek, environmentaverage_ek, Pr.Ek);
	//fprintf(stderr, "NN %e %e \n", _vcoeff, environment_vcoeff);
	
	d_setSinVelocityEntire << < Padd.bloks, SMEMDIM >> > (P0.d_RV0, Padd.d_ER_V, P.d_VU, P.N, x0, 
		float(average_vx), float(average_vy), float(vcoeff), float(tcoeff), float(rcoeff));

	hiprandGenerateNormal(Padd.gen, Padd.d_ER_V, 2 * P.N, 0.0f, sqrtf(1.0f * Po._1d_m * Pr.Ek));
	   
	d_getSinVelocityEntire <<<Padd.bloks4, SMEMDIM >> > (P0.d_RV0, Padd.d_ER_V,Padd.d_bD4, P.N);

	HANDLE_ERROR(hipMemcpy(Padd.h_bD4, Padd.d_bD4, 4 * Padd.bloks4 * sizeof(float), hipMemcpyDeviceToHost));
	average_vx = 0;
	average_vy = 0;
	average_ek = 0;
	n = 0;	
	for (int i = 0; i < Padd.bloks4; ++i)
	{
		average_vx += Padd.h_bD4[i];
		average_vy += Padd.h_bD4[i + Padd.bloks4];
		average_ek += Padd.h_bD4[i + 2 * Padd.bloks4];
		n += Padd.h_bD4[i + 3 * Padd.bloks4];		
		//fprintf(stderr, "n %i %i %e\n", i, bloks, vsh[4][i + bloks]);
	}
	average_vx /= n;
	average_vy /= n;
	average_ek *= 0.5 * Po.m / n;
	average_ek -= 0.5f * Po.m * (average_vx * average_vx + average_vy * average_vy);
	vcoeff = sqrt(Pr.Ek / average_ek);
	tcoeff = Pr.EkSpot / Pr.Ek;
	rcoeff = 2.0 * MC_pi / L.PS[0].x;
	
	//fprintf(stderr, "N %e %e %i | %e %e %e %e | %e %e | %e %e\n", _n, environment_n, P.N,
	//	average_vx, average_vy, environmentaverage_vx, environmentaverage_vy,
	//	average_ek, Pr.Ek, environmentaverage_ek, Pr.Ek);
	//fprintf(stderr, "NN %e %e \n", _vcoeff, environment_vcoeff);
	
	d_setSinVelocityEntire <<< Padd.bloks, SMEMDIM >> > (P0.d_RV0, Padd.d_ER_V,	P.d_VV, P.N, x0,
		float(average_vx), float(average_vy), float(vcoeff), float(tcoeff), float(rcoeff));
}

