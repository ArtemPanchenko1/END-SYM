#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>
//#include <cudpp.h>
//#include <cudpp_plan.h>
__global__ void d_calculateIncrementsFIRE(const float* __restrict__ _1d_Mass, const float* __restrict__ F, float* __restrict__ V, float* __restrict__ U,
	const unsigned int n, const float P_dt, const float P_vis, const float F_alpha)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float vx, vy, v, f, v_d_f;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//if(blockIdx.x > 3)printf("Inc %u %u %u %u %u\n", idx, n, threadIdx.x, blockIdx.x, blockDim.x);
	if (idx < n)
	{
		//printf("In %e %e %e %e %e %e\n", F[idx], F[idx + n], V[idx], V[idx + n], _1d_Mass[idx], P_dt);
		/*if (idx == 700)
			{
				//Fetx[nt] = F[700];
				//Fety[nt] = F[700+n];

				//printf("F00 %e %e\n", F[500], F[500 + n]);
				//printf("F0 %e %e\n", F[990], F[990+n]);
				//printf("F1 %e %e\n", F[995], F[995 + n]);
				//printf("F2 %e %e\n", F[999], F[999 + n]);
			}/**/
			//if(F[idx] > 0)printf("F %i %e %e\n", idx, F[idx], F[idx + n]);
			//if(V[idx]>0)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);

			//if(idx>76648 && idx<76771)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);
	//Leapfrog
#ifndef pre_Viscocity
		vx = V[idx] + F[idx] * _1d_Mass[idx] * P_dt;
		vy = V[idx + n] + F[idx + n] * _1d_Mass[idx] * P_dt;
#endif // !pre_Viscocity		
#ifdef pre_Viscocity
		vx = (F[idx] * _1d_Mass[idx] - P_vis * V[idx]) * P_dt;
		vy = (F[idx + n] * _1d_Mass[idx] - P_vis * V[idx + n]) * P_dt;
#endif // pre_Viscocity

		v = vx * vx + vy * vy;
		f = F[idx] * F[idx] + F[idx + n] * F[idx + n];
		if (f > 1e-12)
		{
			v_d_f = __fsqrt_rn(v * __frcp_rn(f));
		}
		else
		{
			v_d_f = __fsqrt_rn(v)*1e6;
		}
			vx = (1.0f - F_alpha) * vx + F_alpha * v_d_f * F[idx];
			vy = (1.0f - F_alpha) * vy + F_alpha * v_d_f * F[idx + n];
		
		//if (idx == 700)printf("F %i %e %e %e | %e %e\n", idx, v, f, v_d_f, vx, vy);
		

		V[idx] = vx;
		V[idx + n] = vy;
		//V[idx] += (F[idx] * _1d_Mass[idx] - P_vis * V[idx]) * P_dt;
		//V[idx + n] += (F[idx + n] * _1d_Mass[idx] - P_vis * V[idx + n]) * P_dt;
		//if (V[idx]* V[idx] + V[idx + n] * V[idx + n] > 1e-5)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);
		U[idx] += vx * P_dt;
		U[idx + n] += vy * P_dt;
	}
}

__global__ void d_calculateDecrementsHalfStepFIRE(const float* __restrict__ V, float* __restrict__ U, const unsigned int n, const float P_dt)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	if (idx < n)
	{		
	//Leapfrog
		U[idx] -= V[idx] * P_dt * 0.5;
		U[idx + n] -= V[idx + n] * P_dt * 0.5;
	}
}

void calculateGPUStepsContractRelaxFIRE(p_data& P, p0_data& P0, potential_data& Po, pAdd_data& Padd, l_data& L, pNet_data& Pnet, firerelax_data &Fire)
{
	//std::cerr << "W1\n";
	/*hipEvent_t start, stop;
	float gpuTime;
	double gpuTimeAver = 0;/**/
	//float dV, maxShift = 10.0, Vl = sqrt(Po.a * Po.a * Po.c * Po._1d_m), Vd = 1.01;
	///Padd.V = 2e-2;
	char filename[256] = "";
	unsigned int time, bloks, estep, Estep, esize = Padd.ElementSteps * ResultFRNum * (P.NBPT[0] + P.NBPT[1]), i, j, steps;// , timestart, t1, t2;
	double d = 0, povis0 = Po.vis, timereal=0;
	float v = Padd.V, fa_max, fb_min, Ep0 = 2.21821;
	bool contraction = true;


	time = Padd.time;
	//timestart = 1.01*(2.0 * Padd.MaxShift / Padd.V) / Po.dt + 1;

	//float v, v2;
	//time = 1500000;
	
	Fire.bloks4 = P.N / (4 * SMEMDIM) + 1;
	Fire.h_FdotV = (float*)malloc(Fire.bloks4 * sizeof(float));
	HANDLE_ERROR(hipMalloc((void**)&Fire.d_FdotV, Fire.bloks4 * sizeof(float)));
	Padd.bloks = P.N / (SMEMDIM)+2;
	Padd.bloksb = P.NBP / (SMEMDIM)+2;

	std::cerr << "Bloks " << Padd.bloks << " " << Padd.bloksb << "\n";
#ifdef pre_OneNodeContractRelax
	double rMoveOne = 1.0 * 0.01 * 0.03987475 * ReadCoordinatesCoefficient, v1;
	unsigned int onecontracttime = unsigned int(rMoveOne / (v * Po.dt));
	v1 = rMoveOne / (onecontracttime * Po.dt);
	std::cerr << "Time_OCC " << onecontracttime << " " << rMoveOne << " " << v1 << "\n";
	for (steps = 0; steps < onecontracttime; ++steps)
	{
		estep = 0;
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
		//std::cerr << "Q2\n";
#ifdef pre_OneNodeContract
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v1, P.iBP[0]);
#endif // pre_OneNodeContract
		d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);

		//if (steps % 100000 == 0)std::cerr << "steps " << steps << "\n";
		/*if (steps % 10000 == 0)
		{
			end = std::chrono::high_resolution_clock::now();
			dr = end - begin;
			std::cerr << "Fin Step" << steps << " " << v << " " << d << " | " << std::chrono::duration_cast<std::chrono::milliseconds>(dr).count() <<" ms " << "\n";
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);
			begin = std::chrono::high_resolution_clock::now();
			std::cin.get();
			//sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
			//SaveLammpsDATAParticles(P, P0, Po, Pnet, filename);
		}/**/

}
#ifdef pre_SaveLammps
	//if (steps % 100000 == 0)
	{
		std::cerr << "Fin Step" << steps << " " << v << " " << d << "\n";
		d_calculateForcesIBound << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, Padd.d_Fbound, P.N, P.NI, Po.c);


		hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(Padd.h_Fbound, Padd.d_Fbound, P.NI * sizeof(float), hipMemcpyDeviceToHost);
		//sprintf(filename, "./result/steps/CP_%li.txt", steps);
		//SaveTXTParticles(P, P0, Po, Pnet, filename);
		sprintf(filename, "./result/steps/LAMMPS/CP_%li.txt", steps);
		SaveLammpsDATAParticles(P, P0, Po, Pnet, Padd, filename);
	}
#endif // pre_SaveLammps
#endif // pre_OneNodeContractRelax

#ifdef pre_CalcFullEnergy
	sprintf(filename, "./result/CP_Energy_FIRE.txt");
	std::ofstream file_energy_txt;
	file_energy_txt.open(filename, std::ios::out);
	file_energy_txt << "step time Ep Ek Efull\n";
	file_energy_txt.precision(10);
#endif // pre_CalcFullEnergy
	/*hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	sprintf(filename, "./result/steps/CP_%li.txt", 0);
	SaveTXTParticles(P, P0, Po, Pnet, filename);/**/

	unsigned int contracttime = unsigned int(Padd.RMove / (Padd.V * Po.dt));
	v = Padd.RMove / (contracttime * Po.dt);
	std::cerr << "Time_OCC " << contracttime << " " << Padd.RMove << " " << v << " " << Padd.V << " " << Po.dt << "\n";
	timereal = 0;
	for (steps = 0; steps < contracttime; ++steps)
	{
		estep = 0;
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
#ifndef pre_MoveCell
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // !pre_MoveCell
#ifdef pre_MoveCell
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // pre_MoveCell	
		d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);
		timereal += Po.dt;
		/*if (steps % 1000 == 0)
		{			
			std::cerr << "Fin Step" << steps << " " << v << "\n";
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);
			//std::cin.get();
		}/**/
	}

	/*hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	sprintf(filename, "./result/steps/CP_%li.txt", 1);
	SaveTXTParticles(P, P0, Po, Pnet, filename);/**/

	std::cerr << "FIRE Start\n";
	Fire.NPpositive = 0;
	Fire.NPnegative = 0;
	Fire.dt = Fire.dt0;
	Po.vis = 0;
	Po.vism = Po.vis * Po.m;
	v = 0;
	timereal = 0;
	for (steps = 0; steps < Padd.RelaxationTime; ++steps)
	{
		estep = 0;
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
#ifndef pre_MoveCell
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // !pre_MoveCell
#ifdef pre_MoveCell
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // pre_MoveCell	
		//d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);
		//std::cerr << "B " << Fire.bloks4 << "\n";
		d_FdotVEntire << < Fire.bloks4, SMEMDIM >> > (P.d_V, P.d_F, Fire.d_FdotV, P.N);
		hipMemcpy(Fire.h_FdotV, Fire.d_FdotV, Fire.bloks4 * sizeof(float), hipMemcpyDeviceToHost);
		Fire.FdotV = 0;
		for (i = 0; i < Fire.bloks4; ++i)
		{
			Fire.FdotV += Fire.h_FdotV[i];
		}


		if (Fire.FdotV > 0)
		{
			++Fire.NPpositive;
			Fire.NPnegative = 0;
			if (Fire.NPpositive > Fire.Ndelay)
			{
				Fire.dt = (Fire.dt * Fire.dtgrow < Fire.dtmax) ? Fire.dt * Fire.dtgrow : Fire.dtmax;
				Fire.alpha *= Fire.alphashrink;
			}
			//std::cerr << "FdV POS " << steps <<" "<<Fire.dt << " " << Fire.alpha << " " << Fire.FdotV << "\n";
			//std::cin.get();
		}
		else
		{
			Fire.NPpositive = 0;
			++Fire.NPnegative;
			if (Fire.NPnegative > Fire.NPnegativeMax)
				break;
			if (steps > Fire.Ndelay)
			{
				Fire.dt = (Fire.dt * Fire.dtshrink > Fire.dtmin) ? Fire.dt * Fire.dtshrink : Fire.dtmin;
				Fire.alpha = Fire.alpha0;
			}
			d_calculateDecrementsHalfStepFIRE << < Padd.bloks, SMEMDIM >> > (P.d_V, P.d_U, P.N, Fire.dt);
			hipMemset(P.d_V, 0, 2 * P.N * sizeof(float));
			//std::cerr << "FdV NEG " << steps << " " << Fire.dt << " " << Fire.alpha << " " << Fire.FdotV << "\n";
			//std::cin.get();
		}

#ifdef pre_CalcFullEnergy
		if (steps % 1000 == 0)
		{
			d_getEnergyEntire << < Padd.bloks4, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_1d_IM, P.d_U, P.d_V, Padd.d_Esum, P.N, P.NI, Po.c);
			hipMemcpy(Padd.h_Esum, Padd.d_Esum, 2 * Padd.bloks4 * sizeof(float), hipMemcpyDeviceToHost);
			Padd.Esum[0] = 0;
			Padd.Esum[1] = 0;
			for (i = 0; i < Padd.bloks4; ++i)
			{
				Padd.Esum[0] += Padd.h_Esum[i];
				Padd.Esum[1] += Padd.h_Esum[i + Padd.bloks4];
			}
			file_energy_txt << steps << " " << timereal << " " << Padd.Esum[0] - Ep0 << " " << Padd.Esum[1] << " " << Padd.Esum[0] - Ep0 + Padd.Esum[1] << "\n";
			if (steps % 10000 == 0)
				std::cerr << "E " << Padd.Esum[0] - Ep0 << " " << Padd.Esum[1] << " | " << Fire.FdotV << " | " << Fire.NPpositive << " " << Fire.NPnegative << " | " << Fire.dt << "\n";
			//if (steps == 0)Ep0 = Padd.Esum[0];
			
			
			if (steps % 1000000 == 0)
			{
				hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
				sprintf(filename, "./result/steps/CP_%li.txt", steps);
				SaveTXTParticles(P, P0, Po, Pnet, filename);
			}/**/
			
			//if (steps % 2000000 == 0)
			//	std::cin.get();
		}/**/
#endif // pre_CalcFullEnergy
		d_calculateIncrementsFIRE << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Fire.dt, Po.vis, Fire.alpha);
		
		/*if (steps % 1000 == 0)
		{
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);
			//std::cin.get();
		}/**/
		if (steps % 1000000 == 0)
			std::cerr << "Fire relaxation " << steps << "\n";
		timereal += Fire.dt;
		//std::cin.get();
	}	
	std::cerr << "FIN FIRE! " << steps << "\n"; //std::cin.get();
	Po.vis = 0;
	Po.vism = Po.vis * Po.m;
	hipMemset(P.d_V, 0, 2 * P.N * sizeof(float));
	v = 0;
	for (unsigned int steps = 0; steps < Padd.time; ++steps)
	{

		estep = steps % Padd.ElementSteps;
		d_calculateForcesI << < Padd.bloks, SMEMDIM >> > (P.d_In, P.d_Ir0, P.d_ShIn, P.d_U, P.d_F, P.N, P.NI, Po.c);
#ifndef pre_MoveCell
		d_calculateBorders << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // !pre_MoveCell
#ifdef pre_MoveCell
		d_calculateBordersMove << < Padd.bloksb, SMEMDIM >> > (P.d_1d_IM, P.d_BP, P.d_BPR, P.d_F, P.d_V, P.d_U, Padd.d_FResult, P.N, P.NBP, P.NBPT[0], P.NBPT[1], P.NBPT[2], estep, Po.vis, v);
#endif // pre_MoveCell	
		d_calculateIncrements << < Padd.bloks, SMEMDIM >> > (P.d_1d_IM, P.d_F, P.d_V, P.d_U, P.N, Po.dt, Po.vis);
		/*if (estep == 0 && steps > 0)
		{
			hipMemcpy(Padd.h_FResult + Estep * esize, Padd.d_FResult, esize * sizeof(float), hipMemcpyDeviceToHost);
			std::cerr << "AAA!\n";
			++Estep;
		}/**/
		/*if (steps % 10 == 0)
		{
			hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
			sprintf(filename, "./result/steps/CP_%li.txt", steps);
			SaveTXTParticles(P, P0, Po, Pnet, filename);
			std::cin.get();
		}/**/
		//std::cin.get();
	}
	/*hipMemcpy(P.h_U, P.d_U, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(P.h_V, P.d_V, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(P.h_F, P.d_F, 2 * P.N * sizeof(float), hipMemcpyDeviceToHost);
	sprintf(filename, "./result/steps/CP_%li.txt", steps);
	SaveTXTParticles(P, P0, Po, Pnet, filename);/**/
	//std::cerr << "AA! " << Padd.h_FResult << " " << Padd.d_FResult << " " << Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << "\n";
	hipMemcpy(Padd.h_FResult, Padd.d_FResult, Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) * sizeof(float), hipMemcpyDeviceToHost);
	//std::cerr << "AA! " << Padd.h_FResult << " " << Padd.d_FResult << " " << Padd.time * ResultFRNum * (P.NBPT[0] + P.NBPT[1]) << "\n";
	//std::cerr << "AA! " << Padd.h_FResult[0] << " " << Padd.h_FResult[100] << " " << Padd.h_FResult[1000] << "\n";
	//std::cerr << "AAA!\n";
	calculate_Faver2(Padd, P, P0, Po);
	//calculate_Fminmax2(Padd, P, P0, Po);
	hipMemset(P.d_V, 0, 2 * P.N * sizeof(float));


	//hipMemcpy(Padd.h_Fstx, Padd.d_Fstx, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fsty, Padd.d_Fsty, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fetx, Padd.d_Fety, time * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(Padd.h_Fety, Padd.d_Fety, time * sizeof(float), hipMemcpyDeviceToHost);
	//SaveTXTGraphsFR(Padd, P, P0, Po);

	//std::cin.get();/**/

	//HANDLE_ERROR(hipMemcpy(Padd.h_Ek, Padd.d_Ek, (NGPUEk * 4 * P0.N * sizeof(float)), hipMemcpyDeviceToHost));
	//HANDLE_ERROR(hipMemset((void*)Padd.d_Ek, 0, (NGPUEk * 4 * P0.N * sizeof(float))));
#ifdef pre_CalcFullEnergy	
	file_energy_txt.close();
#endif // pre_CalcFullEnergy

	free(Fire.h_FdotV);
	Fire.h_FdotV = nullptr;
	hipFree(Fire.d_FdotV);
	Fire.d_FdotV = nullptr;
}


__global__ void d_FdotVEntire(const float* __restrict__ V, const float* __restrict__ F, float* FdotV, const unsigned int n)
{
	// static shared memory
	__shared__ float s_mem[SMEMDIM];

	// set thread ID
	// global index, 4 blocks of input data processed at a time
	unsigned int tid = threadIdx.x, idx = blockIdx.x * blockDim.x * 4 + threadIdx.x, i;	
	// unrolling 4 blocks
	float fdv = 0;

	// boundary check
	if (idx + 3 * blockDim.x < n)
	{
		float t_fdv0 = 0, t_fdv1 = 0, t_fdv2 = 0, t_fdv3 = 0;
		i = idx;
		t_fdv0 = F[i] * V[i] + F[i + n] * V[i + n];
		i = idx + blockDim.x;
		t_fdv1 = F[i] * V[i] + F[i + n] * V[i + n];
		i = idx + 2 * blockDim.x;
		t_fdv2 = F[i] * V[i] + F[i + n] * V[i + n];
		i = idx + 3 * blockDim.x;
		t_fdv3 = F[i] * V[i] + F[i + n] * V[i + n];
		fdv = t_fdv0 + t_fdv1 + t_fdv2 + t_fdv3;
	}
	else if (idx + 2 * blockDim.x < n)
	{
		float t_fdv0 = 0, t_fdv1 = 0, t_fdv2 = 0;
		i = idx;
		t_fdv0 = F[i] * V[i] + F[i + n] * V[i + n];
		i = idx + blockDim.x;
		t_fdv1 = F[i] * V[i] + F[i + n] * V[i + n];
		i = idx + 2 * blockDim.x;
		t_fdv2 = F[i] * V[i] + F[i + n] * V[i + n];		
		fdv = t_fdv0 + t_fdv1 + t_fdv2;
	}
	else if (idx + blockDim.x < n)
	{
		float t_fdv0 = 0, t_fdv1 = 0;
		i = idx;
		t_fdv0 = F[i] * V[i] + F[i + n] * V[i + n];
		i = idx + blockDim.x;
		t_fdv1 = F[i] * V[i] + F[i + n] * V[i + n];		
		fdv = t_fdv0 + t_fdv1;
	}
	else if (idx < n)
	{
		float t_fdv0 = 0;
		i = idx;
		t_fdv0 = F[i] * V[i] + F[i + n] * V[i + n];		
		fdv = t_fdv0;
	}/**/

	//if(idx + 5 * blockDim.x >4619700)
	//   printf("TT %i %i %i %f %i\n", tid, idx, blockIdx.x, n);
	//if (ns>1e-3f)
	//   printf("TT %i %i %f\n", tid, idx, ns);
	s_mem[tid] = fdv;
	__syncthreads();

	//if(idx==0)
	//	printf("TT %i %f %f %i %i\n", tid, s_ek, e_ek, s_n, e_n);

	// in-place reduction in shared memory
	if (blockDim.x >= 1024 && tid < 512)
	{
		s_mem[tid] += s_mem[tid + 512];
	}
	__syncthreads();

	if (blockDim.x >= 512 && tid < 256)
	{
		//printf("Blok!\n");
		s_mem[tid] += s_mem[tid + 256];
	}
	__syncthreads();

	if (blockDim.x >= 256 && tid < 128)
	{
		s_mem[tid] += s_mem[tid + 128];
	}
	__syncthreads();

	if (blockDim.x >= 128 && tid < 64)
	{
		s_mem[tid] += s_mem[tid + 64];
	}

	__syncthreads();
	/*if (blockIdx.x == 0 && threadIdx.x == 0)
	{
		for (int i = 0; i < SMEMDIM; ++i)
			printf("GM %i %e\n", i, smem[i + 3 * SMEMDIM]);
	}/**/

	// unrolling warp
	if (tid < 32)
	{
		volatile float* vsmem = s_mem;
		vsmem[tid] += vsmem[tid + 32];		
		vsmem[tid] += vsmem[tid + 16];		
		vsmem[tid] += vsmem[tid + 8];		
		vsmem[tid] += vsmem[tid + 4];		
		vsmem[tid] += vsmem[tid + 2];		
		vsmem[tid] += vsmem[tid + 1];		
	}/**/

	// write result for this block to global mem
	if (tid == 0)
	{
		//printf("TT %i %i %i %f\n", tid, idx, blockIdx.x, 0);
		FdotV[blockIdx.x] = s_mem[0];
		//printf("TTT %i %i %i %f\n", tid, idx, blockIdx.x, FdotV[blockIdx.x]);
		//if (smem[tid + 3 * SMEMDIM] > 1e-3f)
		//	printf("TT %i %i %f\n", tid, idx, smem[tid + 3 * SMEMDIM]);
		//if (smem[3 * SMEMDIM] > 1e-3f)
		//printf("T %i %f\n", blockIdx.x, gridDim.x, smem[3 * SMEMDIM]);
	}/**/
}