#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>


__global__ void d_calculateBordersMove(const float* __restrict__ _1d_Mass, const int* __restrict__ BP, const float* __restrict__ BPR, float* __restrict__ F, float* __restrict__ V, const float* __restrict__ U, float* __restrict__ FR,
	const unsigned int n, const unsigned int nbp, const unsigned int nbpt1, const unsigned int nbpt2, const unsigned int nbpt3, const unsigned int Step, const float P_vis, const float V1)
{

	__shared__ float FA[2*SMEMDIM];
	__shared__ float FB[2*SMEMDIM];
#ifdef pre_RotateCell
	__shared__ float MA[2*SMEMDIM];
	__shared__ float MB[SMEMDIM];
#endif // pre_RotateCell

	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x, i, j, iA, iB;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//printf("B %i %i\n", idx, nbp);
	float rx, ry, _1d_rm, rux, ruy, rum, m, ax, ay, vtx, vty, vr;
	if (idx < nbp)
	{
		i = BP[idx];
		iA = n - 2;
		iB = n - 1;
		if (idx < nbpt1)
		{			
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			rx = BPR[nbpt1 - 1] - BPR[idx] + U[iA] - U[i];
			ry = BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp] + U[iA + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			FA[idx] = F[i];
			FA[idx + nbpt1] = F[i + n];
#ifdef pre_RotateCell
			MA[idx] = rx * F[i + n] - ry * F[i];
			MA[idx + nbpt1] = __fsqrt_rn(rx * rx + ry * ry);
			//printf("A0 %u %u | %e\n", idx, Step, MA[idx + nbpt1]);
#endif // pre_RotateCell
			//printf("FA %i | %e %e | %i | %e %e\n", idx, F[i], F[i + n], ResultFRNum * Step * (nbpt1 + nbpt2), FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)]);
			//printf("B0 %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, _1d_rm, BPR[idx], BPR[idx + nbp]);
		}
		else if (idx < nbpt1 + nbpt2)
		{			
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			//printf("FAB  %i %i | %i %i | %e %e\n", idx, ResultFRNum, Step, nbpt1 + nbpt2, F[i], F[i + n]);			
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			FB[idx - nbpt1] = F[i];
			FB[idx - nbpt1 + nbpt2] = F[i + n];
#ifdef pre_RotateCell
			MB[idx - nbpt1] = rx * F[i + n] - ry * F[i];
#endif // pre_RotateCell
		}
		else
		{
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;
		}
		if (idx == 0)
		{
			
			for (j = 1; j < nbpt1-1; ++j)
			{
				FA[0] += FA[j];
				FA[0 + nbpt1] += FA[j + nbpt1];
#ifdef pre_RotateCell
				MA[0] += MA[j];
				MA[nbpt1] += MA[j + nbpt1];
#endif // pre_RotateCell
			}
			//FA[0] = 0; FA[0 + nbpt1] = 0; V[iA] = 0; V[iA + n] = 0;
			F[iA] = FA[0];
			F[iA + n] = FA[0 + nbpt1];
			FA[0] *= _1d_Mass[iA];
			FA[0 + nbpt1] *= _1d_Mass[iA];
#ifdef pre_RotateCell			
			MA[nbpt1] = __fdiv_rn(float(nbpt1), MA[nbpt1]);
			MA[0] *= 2.0f * _1d_Mass[iA] * MA[nbpt1] * MA[nbpt1];	
			//printf("AA %u %u | %e %e\n", idx, Step, MA[nbpt1], MA[0]);
#endif // pre_RotateCell
			//printf("AA %i %i\n", idx, Step);
			
//F[iA] = 0;	F[iA + n] = 0;	V[iA] = 0;	V[iA + n] = 0;
			//printf("FAB  %e %e | %e %e\n", FA[0], FB[0], FA[0 + nbpt1], FB[0 + nbpt2]);
		}
		else if (idx == 1)
		{
			for (j = 1; j < nbpt2 - 1; ++j)
			{
				FB[0] += FB[j];
				FB[0 + nbpt2] += FB[j + nbpt2];
#ifdef pre_RotateCell
				MB[0] += MB[j];
				//printf("BB %i | %e %e\n", j, MB[0], MB[j]);
#endif // pre_RotateCell
			}
			//FB[0] = 0; FB[0 + nbpt2] = 0; V[iB] = 0; V[iB + n] = 0;
			F[iB] = FB[0];
			F[iB + n] = FB[0 + nbpt2];
			FB[0] *= _1d_Mass[iB];
			FB[0 + nbpt2] *= _1d_Mass[iB];
#ifdef pre_RotateCell
			MB[0] *= 2.0f * _1d_Mass[iB] * 628.93252341f;
			//printf("BB %u %u | %e\n", idx, Step, MB[0]);
#endif // pre_RotateCell
//F[iB] = 0;	F[iB + n] = 0;	V[iB] = 0;	V[iB + n] = 0;
		}
		__syncthreads();
		if (idx < nbpt1-1)
		{

			
			//if(Step%10000==0)printf("A %i %i | %e %e | \%e %e\n", idx, Step, FR[idx + (ResultFRNum * Step + 0) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)]);
			//printf("A %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
			_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
			vtx = V[i] - V[iA];
			vty = V[i + n] - V[iA + n];
			vr = (vtx * rx + vty * ry) * _1d_rm * _1d_rm;
			vtx -= vr * rx;
			vty -= vr * ry;
			//printf("A %i | %e %e | %e %e | %e\n", idx, rx, ry, vtx, vty, rx*vtx+ry*vty);
			V[i] = V[iA] + V1 * rx * _1d_rm + vtx;
			V[i + n] = V[iA + n] + V1 * ry * _1d_rm + vty;
			m = __frcp_rn(_1d_Mass[i]);
			ax = FA[0];
			ay = FA[0 + nbpt1];
#ifdef pre_RotateCell
			ax -= MA[0] * ry;
			ay += MA[0] * rx;
#endif // pre_RotateCell
#ifdef pre_Relaxation
			ax += P_vis * vr * rx;
			ay += P_vis * vr * ry;
#endif // pre_Relaxation
#ifdef pre_Viscocity
			ax += P_vis * V1 * rx * _1d_rm;
			ay += P_vis * V1 * ry * _1d_rm;
#endif // pre_Relaxation
			F[i] = ax * m;
			F[i + n] = ay * m;
			//F[i] = 0;
			//F[i + n] = 0;
//F[i] = 0;	F[i + n] = 0;
			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, _1d_rm, BPR[idx], BPR[idx + nbp]);
		} else if (idx >= nbpt1 && idx < nbpt1 + nbpt2 - 1)
		{
			
			//if (Step % 10000 == 0)printf("B %i %i | %e %e | \%e %e\n", idx, Step, FR[idx + (ResultFRNum * Step + 0) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)]);
			_1d_rm = __frcp_rn(rx * rx + ry * ry);
			vtx = V[i] - V[iB];
			vty = V[i + n] - V[iB + n];
			vr = (vtx * rx + vty * ry) * _1d_rm;
			vtx -= vr * rx;
			vty -= vr * ry;
			V[i] = V[iB] + vtx;
			V[i + n] = V[iB + n] + vty;
			//printf("B %i | %e %e | %e %e | %e\n", idx, rx, ry, vtx, vty, rx* vtx + ry * vty);
			//V[i] = V[iB] + vtx;
			//V[i + n] = V[iB + n] + vty;
			m = __frcp_rn(_1d_Mass[i]);
			ax = FB[0];
			ay = FB[0 + nbpt2];
#ifdef pre_RotateCell
			ax -= MB[0] * ry;
			ay += MB[0] * rx;
#endif // pre_RotateCell
			F[i] = ax * m;
			F[i + n] = ay * m;
//F[i] = 0;	F[i + n] = 0;	V[i] = 0;	V[i + n] = 0;
		}
	}	
}

__global__ void d_calculateBordersMove(const float* __restrict__ _1d_Mass, const int* __restrict__ BP, const float* __restrict__ BPR, float* __restrict__ F, float* __restrict__ V, const float* __restrict__ U, float* __restrict__ FR,
	const unsigned int n, const unsigned int nbp, const unsigned int nbpt1, const unsigned int nbpt2, const unsigned int nbpt3, const unsigned int Step, const float P_vis, const float V1, const int ibp)
{
	__shared__ float FA[2 * SMEMDIM];
	__shared__ float FB[2 * SMEMDIM];
#ifdef pre_RotateCell
	__shared__ float MA[2 * SMEMDIM];
	__shared__ float MB[SMEMDIM];
#endif // pre_RotateCell
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x, i, j, iA, iB, iibp;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//printf("B %i %i\n", idx, nbp);
	float rx, ry, _1d_rm, rux, ruy, rum, m, ax, ay, vtx, vty, vr;
	if (idx < nbp)
	{
		i = BP[idx];
		iibp = BP[ibp];
		iA = n - 2;
		iB = n - 1;
		if (idx < nbpt1)
		{
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			rx = BPR[nbpt1 - 1] - BPR[idx] + U[iA] - U[i];
			ry = BPR[nbpt1 - 1 + nbp] - BPR[idx + nbp] + U[iA + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			FA[idx] = F[i];
			FA[idx + nbpt1] = F[i + n];
#ifdef pre_RotateCell
			MA[idx] = rx * F[i + n] - ry * F[i];
			MA[idx + nbpt1] = __fsqrt_rn(rx * rx + ry * ry);
			//printf("A0 %u %u | %e\n", idx, Step, MA[idx + nbpt1]);
#endif // pre_RotateCell
			//printf("FA %i | %e %e | \%e %e\n", idx, F[i], F[i + n]);
			//printf("B0 %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, _1d_rm, BPR[idx], BPR[idx + nbp]);
		}
		else if (idx < nbpt1 + nbpt2)
		{
			FR[idx + ResultFRNum * Step * (nbpt1 + nbpt2)] = F[i];
			FR[idx + (ResultFRNum * Step + 1) * (nbpt1 + nbpt2)] = F[i + n];
			rx = BPR[nbpt1 + nbpt2 - 1] - BPR[idx] + U[iB] - U[i];
			ry = BPR[nbpt1 + nbpt2 - 1 + nbp] - BPR[idx + nbp] + U[iB + n] - U[i + n];
			FR[idx + (ResultFRNum * Step + 2) * (nbpt1 + nbpt2)] = rx;
			FR[idx + (ResultFRNum * Step + 3) * (nbpt1 + nbpt2)] = ry;
			//printf("FAB  %i %i | %i %i | %e %e\n", idx, ResultFRNum, Step, nbpt1 + nbpt2, F[i], F[i + n]);
			FB[idx - nbpt1] = F[i];
			FB[idx - nbpt1 + nbpt2] = F[i + n];
#ifdef pre_RotateCell
			MB[idx - nbpt1] = rx * F[i + n] - ry * F[i];
#endif // pre_RotateCell
		}
		else
		{
			F[i] = 0;
			F[i + n] = 0;
			V[i] = 0;
			V[i + n] = 0;
		}
		if (idx == 0)
		{

			for (j = 1; j < nbpt1 - 1; ++j)
			{
				FA[0] += FA[j];
				FA[0 + nbpt1] += FA[j + nbpt1];
#ifdef pre_RotateCell
				MA[0] += MA[j];
				MA[nbpt1] += MA[j + nbpt1];
#endif // pre_RotateCell
			}
			//FA[0] = 0; FA[0 + nbpt1] = 0; V[iA] = 0; V[iA + n] = 0;
			F[iA] = FA[0];
			F[iA + n] = FA[0 + nbpt1];
			FA[0] *= _1d_Mass[iA];
			FA[0 + nbpt1] *= _1d_Mass[iA];
#ifdef pre_RotateCell			
			MA[nbpt1] = __fdiv_rn(float(nbpt1), MA[nbpt1]);
			MA[0] *= 2.0f * _1d_Mass[iA] * MA[nbpt1] * MA[nbpt1];
			//printf("AA %u %u | %e %e\n", idx, Step, MA[nbpt1], MA[0]);
#endif // pre_RotateCell
			//F[iA] = 0;	F[iA + n] = 0;	V[iA] = 0;	V[iA + n] = 0;
						//printf("FAB  %e %e | %e %e\n", FA[0], FB[0], FA[0 + nbpt1], FB[0 + nbpt2]);
		}
		else if (idx == 1)
		{
			for (j = 1; j < nbpt2 - 1; ++j)
			{
				FB[0] += FB[j];
				FB[0 + nbpt2] += FB[j + nbpt2];
#ifdef pre_RotateCell
				MB[0] += MB[j];
				//printf("BB %i | %e %e\n", j, MB[0], MB[j]);
#endif // pre_RotateCell
			}
			//FB[0] = 0; FB[0 + nbpt2] = 0; V[iB] = 0; V[iB + n] = 0;
			F[iB] = FB[0];
			F[iB + n] = FB[0 + nbpt2];
			FB[0] *= _1d_Mass[iB];
			FB[0 + nbpt2] *= _1d_Mass[iB];
#ifdef pre_RotateCell
			MB[0] *= 2.0f * _1d_Mass[iB] * 628.93252341f;
			//printf("BB %u %u | %e\n", idx, Step, MB[0]);
#endif // pre_RotateCell
			//F[iB] = 0;	F[iB + n] = 0;	V[iB] = 0;	V[iB + n] = 0;
		}		
		__syncthreads();
		if (idx < nbpt1 - 1)
		{

			_1d_rm = __frsqrt_rn(rx * rx + ry * ry);
			vtx = V[i] - V[iA];
			vty = V[i + n] - V[iA + n];
			vr = (vtx * rx + vty * ry) * _1d_rm * _1d_rm;
			vtx -= vr * rx;
			vty -= vr * ry;
			
			//printf("B %i | %e %e | \%e %e\n", idx, rum, rux * rux + ruy * ruy, rx, ry);
			ax = FA[0];
			ay = FA[0 + nbpt1];
			if (idx == ibp)
			{				
				V[i] = V[iA] + V1 * rx * _1d_rm + vtx;
				V[i + n] = V[iA + n] + V1 * ry * _1d_rm + vty;
#ifdef pre_Relaxation
				ax += P_vis * V1 * rx * _1d_rm;
				ay += P_vis * V1 * ry * _1d_rm;
#endif // pre_Relaxation
#ifdef pre_Viscocity
				ax += P_vis * vr * rx;
				ay += P_vis * vr * ry;
#endif // pre_Relaxation
			}
			else
			{
				V[i] = V[iA] + vtx;
				V[i + n] = V[iA + n] + vty;
			}
			m = __frcp_rn(_1d_Mass[i]);
#ifdef pre_RotateCell
			ax -= MA[0] * ry;
			ay += MA[0] * rx;
#endif // pre_RotateCell
			F[i] = ax * m;
			F[i + n] = ay * m;
//F[i] = 0;	F[i + n] = 0;
			//printf("F %i %i | %e %e | %e %e | %e %e %e | %e %e\n", idx, i, F[i], F[i + n], V[i], V[i + n], rx, ry, _1d_rm, BPR[idx], BPR[idx + nbp]);
		}
		else if (idx >= nbpt1 && idx < nbpt1 + nbpt2 - 1)
		{
			_1d_rm = __frcp_rn(rx * rx + ry * ry);
			vtx = V[i] - V[iB];
			vty = V[i + n] - V[iB + n];
			vr = (vtx * rx + vty * ry) * _1d_rm;
			vtx -= vr * rx;
			vty -= vr * ry;
			V[i] = V[iB] + vtx;
			V[i + n] = V[iB + n] + vty;
			//printf("B %i | %e %e | %e %e | %e\n", idx, rx, ry, vtx, vty, rx* vtx + ry * vty);
			//V[i] = V[iB] + vtx;
			//V[i + n] = V[iB + n] + vty;
			m = __frcp_rn(_1d_Mass[i]);
			ax = FB[0];
			ay = FB[0 + nbpt2];
#ifdef pre_RotateCell
			ax -= MB[0] * ry;
			ay += MB[0] * rx;
#endif // pre_RotateCell
			F[i] = ax * m;
			F[i + n] = ay * m;
			//F[i] = 0;	F[i + n] = 0;	V[i] = 0;	V[i + n] = 0;
		}
	}
}

__global__ void d_PrintFR(const float* __restrict__ FR, const unsigned int n, const unsigned int nbp, const unsigned int nbpt1, const unsigned int nbpt2, const unsigned int nbpt3, const unsigned int Step)
{	
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x, i, j, iA, iB;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//printf("B %i %i\n", idx, nbp);
	float rx, ry, _1d_rm, rux, ruy, rum, m;
	//printf("FR %i %i\n", idx, Step);
	if (idx == 0)
	{
		//printf("FR %i %i\n", idx, Step);
		//for (j = 0; j < Step; ++j)
		j = Step;
		{
			printf("FR %i | %e %e | %e %e\n", j, ResultFRNum, FR[idx + ResultFRNum * j * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * j + 1) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * j + 2) * (nbpt1 + nbpt2)], FR[idx + (ResultFRNum * j + 3) * (nbpt1 + nbpt2)]);
		}

		
	}		
}