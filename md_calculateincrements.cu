#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>


__global__ void d_calculateIncrements(const float* __restrict__ _1d_Mass, const float* __restrict__ F, float* __restrict__ V, float* __restrict__ U,
	const unsigned int n, const float P_dt, const float P_vis)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//if(blockIdx.x > 3)printf("Inc %u %u %u %u %u\n", idx, n, threadIdx.x, blockIdx.x, blockDim.x);
	if (idx < n)
	{
		/*if (idx == 700)
			{
				//Fetx[nt] = F[700];
				//Fety[nt] = F[700+n];

				//printf("F00 %e %e\n", F[500], F[500 + n]);
				//printf("F0 %e %e\n", F[990], F[990+n]);
				//printf("F1 %e %e\n", F[995], F[995 + n]);
				//printf("F2 %e %e\n", F[999], F[999 + n]);
			}/**/
		//if(F[idx] > 0)printf("F %i %e %e\n", idx, F[idx], F[idx + n]);
		//if(V[idx]>0)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);

		//if(idx>76648 && idx<76771)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);

		//V[idx] += (F[idx] * _1d_Mass[idx] - P_vis * V[idx]) * P_dt;
		//V[idx + n] += (F[idx + n] * _1d_Mass[idx] - P_vis * V[idx + n]) * P_dt;
		V[idx] += F[idx] * _1d_Mass[idx] * P_dt;
		V[idx + n] += F[idx + n] * _1d_Mass[idx] * P_dt;

		//if (V[idx]* V[idx] + V[idx + n] * V[idx + n] > 1e-5)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);

		U[idx] += V[idx] * P_dt;
		U[idx + n] += V[idx + n] * P_dt;
	}	
}

__global__ void d_calculateIncrementsVis(const float* __restrict__ _1d_Mass, const float* __restrict__ VisR, const float* __restrict__ F, float* __restrict__ V, float* __restrict__ U,
	const unsigned int n, const float P_dt, const float P_vis)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float visc;
	//if (idx == 0)printf("In %e %e %e %e |%e %e %e %e | %e %e %e %e | %f %f\n", FV[idx], FV[idx + n], FU[idx], FU[idx + n], VV[idx], VV[idx + n], VU[idx], VU[idx + n], V[idx], V[idx + n], U[idx], U[idx + n], P_dtm, P_dt);
	//if(blockIdx.x > 3)printf("Inc %u %u %u %u %u\n", idx, n, threadIdx.x, blockIdx.x, blockDim.x);
	if (idx < n)
	{
		/*if (idx == 700)
			{
				//Fetx[nt] = F[700];
				//Fety[nt] = F[700+n];

				//printf("F00 %e %e\n", F[500], F[500 + n]);
				//printf("F0 %e %e\n", F[990], F[990+n]);
				//printf("F1 %e %e\n", F[995], F[995 + n]);
				//printf("F2 %e %e\n", F[999], F[999 + n]);
			}/**/
			//if(F[idx] > 0)printf("F %i %e %e\n", idx, F[idx], F[idx + n]);
			//if(V[idx]>0)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);

			//if(idx>76648 && idx<76771)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);
		visc = 6.0f * MCf_pi * P_vis * VisR[idx];
		V[idx] += (F[idx] - visc * V[idx]) * _1d_Mass[idx] * P_dt;
		V[idx + n] += (F[idx + n] - visc * V[idx + n]) * _1d_Mass[idx] * P_dt;

		//if (V[idx]* V[idx] + V[idx + n] * V[idx + n] > 1e-5)printf("V %i %e %e\n", idx, V[idx], V[idx + n]);

		U[idx] += V[idx] * P_dt;
		U[idx + n] += V[idx + n] * P_dt;
	}
}