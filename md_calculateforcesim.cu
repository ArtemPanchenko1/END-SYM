#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>

//__constant__ char IM[2 * IMatrixSize];

__device__ float dd_nonlinearC(const double rmada, const double c)
{
	//if (rmada < -1.0 + 1e-7)std::cerr << "Error!\n";
	if (rmada < -0.9)return c * (0.01 - 1000.1 * (rmada + 0.9));
	if (rmada < -0.1)return c * 0.01;
	if (rmada < 0.0)return c * (0.01 + 9.9 * (rmada + 0.1));
	return c * __expf(2.3025851 * rmada);
}

__global__ void d_calculateForcesI(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, float* __restrict__ F, const unsigned int n, const unsigned int ni, const float P_c)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drx, dry, drm, rmada, _1d_drm, _1d_a, c, fsumx, fsumy;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsumx = 0;
		fsumy = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			drx = Ir0[k] + U[j] - U[i];
			dry = Ir0[k + ni] + U[j + n] - U[i + n];
			_1d_a = Ir0[k + 2 * ni];			
			c = P_c;

#ifdef pre_nonlinearC
			drm = __fsqrt_rn(drx * drx + dry * dry);
			_1d_drm = __frcp_rn(drm);
			rmada = drm * _1d_a - 1.0f;
			c = dd_nonlinearC(rmada, P_c);
#endif
#ifndef pre_nonlinearC
			_1d_drm = __frsqrt_rn(drx * drx + dry * dry);
#endif // !pre_nonlinearC		

			fsumx += c * drx * (_1d_a - _1d_drm);
			fsumy += c * dry * (_1d_a - _1d_drm);
			//if(fsumx* fsumx+ fsumy* fsumy>1e-5)
			//if(idx==4822)
			//	printf("F %i %i %i | %e %e %e | %e %e | %e %e\n", idx, i, j, fsumx, fsumy, _1d_a, U[i], U[i+n], U[j], U[j + n]);
			//printf("FF %e %e %e \n", drx, (1.0 - P_a * _1d_drm), fx[1]);
			//printf("F %u %u | %e %e %e | %e %e | %e %e\n",  j1, idx, fx[1], fy[1], drm, U[j1], U[idx], U[j1 + n], U[idx + n]);

		}
		//if (fsumx * fsumx + fsumy * fsumy > 1e-5)
		
		//if(idx==4818)
		//	printf("F %u | %e %e %e\n", idx, fsumx, fsumy, fsumx * fsumx + fsumy * fsumy);

		F[idx] = fsumx;
		F[idx + n] = fsumy;		
		//if (idx == 4818)	printf("F %u | %e %e\n", idx, F[idx], F[idx + n]);
		idx += blockDim.x * gridDim.x;
	}	
}

__global__ void d_calculateForcesIBound(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, float* __restrict__ Fbound, const unsigned int n, const unsigned int ni, const float P_c)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drx, dry, drm, rmada, _1d_drm, _1d_a, c, fsumx, fsumy;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsumx = 0;
		fsumy = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			drx = Ir0[k] + U[j] - U[i];
			dry = Ir0[k + ni] + U[j + n] - U[i + n];
			_1d_a = Ir0[k + 2 * ni];
			drm = __fsqrt_rn(drx * drx + dry * dry);
			_1d_drm = __frcp_rn(drm);

			c = P_c;

#ifdef pre_nonlinearC
			rmada = drm * _1d_a - 1.0f;
			c = dd_nonlinearC(rmada, P_c);
#endif


			//fsumx = c * drx * (_1d_a - _1d_drm);
			//fsumy = c * dry * (_1d_a - _1d_drm);
			//Fbound[k] = __fsqrt_rn(fsumx * fsumx + fsumy * fsumy);
			Fbound[k] = c * (drm * _1d_a - 1.0f);
			//if(fsumx* fsumx+ fsumy* fsumy>1e-10)
			//printf("F %i %i %i | %e %e | %e %e | %e %e\n", idx, i, j, c * drx * (_1d_a - _1d_drm), c * dry * (_1d_a - _1d_drm), U[i], U[i+n], U[j], U[j + n]);
			//printf("FF %e %e %e \n", drx, (1.0 - P_a * _1d_drm), fx[1]);
			//printf("F %u %u | %e %e %e | %e %e | %e %e\n",  j1, idx, fx[1], fy[1], drm, U[j1], U[idx], U[j1 + n], U[idx + n]);

		}
		//F[idx] = fsumx;
		//F[idx + n] = fsumy;
		idx += blockDim.x * gridDim.x;
	}
}

__global__ void d_calculateEnergyIBound(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, float* __restrict__ Ebound, const unsigned int n, const unsigned int ni, const float P_c)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drx, dry, drm, rmada, _1d_a, c, drmma;//, _1d_drm
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		//fsumx = 0;
		//fsumy = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			drx = Ir0[k] + U[j] - U[i];
			dry = Ir0[k + ni] + U[j + n] - U[i + n];
			_1d_a = Ir0[k + 2 * ni];
			drm = __fsqrt_rn(drx * drx + dry * dry);
			//_1d_drm = __frcp_rn(drm);

			c = P_c;

#ifdef pre_nonlinearC
			rmada = drm * _1d_a - 1.0f;
			c = dd_nonlinearC(rmada, P_c);
#endif

			rmada = drm - __frcp_rn(_1d_a);
			//fsumx = c * drx * (_1d_a - _1d_drm);
			//fsumy = c * dry * (_1d_a - _1d_drm);
			//Fbound[k] = __fsqrt_rn(fsumx * fsumx + fsumy * fsumy);
			Ebound[k] = c * _1d_a * rmada * rmada;
			//if(fsumx* fsumx+ fsumy* fsumy>1e-10)
			//printf("F %i %i %i | %e %e | %e %e | %e %e\n", idx, i, j, c * drx * (_1d_a - _1d_drm), c * dry * (_1d_a - _1d_drm), U[i], U[i+n], U[j], U[j + n]);
			//printf("FF %e %e %e \n", drx, (1.0 - P_a * _1d_drm), fx[1]);
			//printf("F %u %u | %e %e %e | %e %e | %e %e\n",  j1, idx, fx[1], fy[1], drm, U[j1], U[idx], U[j1 + n], U[idx + n]);

		}
		//F[idx] = fsumx;
		//F[idx + n] = fsumy;
		idx += blockDim.x * gridDim.x;
	}
}


__global__ void d_calculateUIBound(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, float* __restrict__ Ubound, const unsigned int n, const unsigned int ni, const float P_c)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float dr;//, _1d_drm
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		//fsumx = 0;
		//fsumy = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];
#ifdef pre_SaveLammpsUx
			dr = 0.5f * (U[j] + U[i]);
			Ubound[k] = dr;
#endif // pre_SaveLammpsUx
		}		
		idx += blockDim.x * gridDim.x;
	}
}