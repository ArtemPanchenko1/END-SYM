#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>
#include <float.h>
#include "m_func.h"

__global__ void seq_minmaxKernel(float* max, float* min, const float* __restrict__ a, const unsigned int n) {
	__shared__ float maxtile[SMEMDIM];
	__shared__ float mintile[SMEMDIM];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
	{
		maxtile[tid] = a[i];
		mintile[tid] = a[i];
	}
	else
	{
		maxtile[tid] = -FLT_MAX;
		mintile[tid] =  FLT_MAX;
	}
		__syncthreads();

		//sequential addressing by reverse loop and thread-id based indexing
		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
			if (tid < s) {
				//printf("FMM %u %u\n", tid, s);
				if (maxtile[tid + s] > maxtile[tid])
					maxtile[tid] = maxtile[tid + s];
				if (mintile[tid + s] < mintile[tid])
					mintile[tid] = mintile[tid + s];
			}
			__syncthreads();
		}
	

	if (tid == 0) {
		//printf("FMaxMin %u | %e %e\n", blockIdx.x, mintile[0], maxtile[0]);
		max[blockIdx.x] = maxtile[0];
		min[blockIdx.x] = mintile[0];
	}
		
}

__global__ void seq_finalminmaxKernel(float* max, float* min, float* __restrict__ mimmax, const unsigned int offset, const unsigned int n) {
	__shared__ float maxtile[SMEMDIM];
	__shared__ float mintile[SMEMDIM];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n)
	{
		maxtile[tid] = max[i];
		mintile[tid] = min[i];
	}
	else
	{
		maxtile[tid] = -FLT_MAX;
		mintile[tid] =  FLT_MAX;
	}
	
	__syncthreads();

	//sequential addressing by reverse loop and thread-id based indexing
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			if (maxtile[tid + s] > maxtile[tid])
				maxtile[tid] = maxtile[tid + s];
			if (mintile[tid + s] < mintile[tid])
				mintile[tid] = mintile[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) {
		//printf("FMaxMin %u | %e %e\n", blockIdx.x, mintile[0], maxtile[0]);
		max[blockIdx.x] = maxtile[0];
		min[blockIdx.x] = mintile[0];
		mimmax[offset] = mintile[0];
		mimmax[offset+1] = maxtile[0];
	}
}