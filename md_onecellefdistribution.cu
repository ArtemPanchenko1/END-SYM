#include "hip/hip_runtime.h"
#include "md_phys_constants.h"
#include "md_data_types.h"
#include "md.h"
#include <stdint.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

__global__ void d_distrubuteBoundToCircle(const int* __restrict__ In, const float* __restrict__ U, const float* __restrict__ RU0, const float* __restrict__ EFbound, const float* __restrict__ EFbound0, float* CEF, const unsigned int n, const unsigned int ni, const unsigned int nT, const double X0, const double Y0, const double _1d_DR)
{
	__shared__ float s_mem[2*NumberCircles];

	int i, j, ir;
	float rcx, rcy, r, EF, uu;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	while (idx < ni)
	{
		//if (idx == 0)printf("F0I %i %i %i\n", idx, ni, n);
		
		if (threadIdx.x < 2*NumberCircles)
			s_mem[threadIdx.x] = 0;

		i = In[idx];
		j = In[idx + ni];
#ifndef pre_OneCellMaterialdistribution		
		rcx = 0.5f * (RU0[i] + RU0[j]) - X0;
		rcy = 0.5f * (RU0[i + n] + RU0[j + n]) - Y0;
#endif // !pre_OneCellMaterialdistribution
#ifdef pre_OneCellMaterialdistribution
		rcx = 0.5f * (RU0[i] + U[i] + RU0[j] + U[j]) - X0;
		rcy = 0.5f * (RU0[i + n] + U[i + n] + RU0[j + n] + U[j + n]) - Y0;
#endif // pre_OneCellMaterialdistribution
		r = __fsqrt_rn(rcx * rcx + rcy * rcy);
		ir = roundf(r * _1d_DR);

		//if (threadIdx.x == 0)
		//	printf("F1I %i %f\n", idx, EFbound[idx]);
		//if(threadIdx.x==0)
		//	printf("F1I %i %i %f | %f %f\n", idx, ir, r, rcx, rcy);
		//if(ir> NumberCircles-1)printf("T %i %f\n", ir, r);

		__syncthreads();
		//if (idx == 0)printf("F0I %i %i %i\n", idx, ni, nT);
		//if (fabsf(ir-10)<0.1)
		{
			/*rcx = 0.5f * (U[i] + U[j]);
			rcy = 0.5f * (U[i + n] + U[j + n]);
			uu = __fsqrt_rn(rcx * rcx + rcy * rcy);
			atomicAdd(s_mem + ir, uu);/**/
			atomicAdd(s_mem + ir, fabsf(EFbound[idx] - EFbound0[idx]));
			//atomicAdd(s_mem + ir, r);
			atomicAdd(s_mem + ir + NumberCircles, 1.0f);
		}
		
		__syncthreads();
		//if (idx == 0 && nT>100)printf("F0I %i %i %i\n", idx, ni, n);
		if (threadIdx.x < NumberCircles-1)
		{
			/*if (s_mem[threadIdx.x + NumberCircles] > 1e-1)
				EF = __fdiv_rn(s_mem[threadIdx.x], s_mem[threadIdx.x + NumberCircles]);
			else
				EF = 0;/**/
			//if(threadIdx.x ==20)
			//	printf("T %i %i %i %i | %e %e\n", idx, 2 * NumberCircles, threadIdx.x, nT, s_mem[threadIdx.x], s_mem[threadIdx.x + NumberCircles]);
			atomicAdd(CEF + 2 * NumberCircles * nT + threadIdx.x, s_mem[threadIdx.x]);
			atomicAdd(CEF + 2 * NumberCircles * nT + NumberCircles + threadIdx.x, s_mem[threadIdx.x + NumberCircles]);
		}
			
		
		__syncthreads();
		//if (idx == 0)
		//	printf("F2I %f\n", CEF[NumberCircles * nT + 10]);/**/
		idx += blockDim.x * gridDim.x;
	}	
}

__global__ void d_calculateForcesI_EF(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, float* __restrict__ F, float* __restrict__ EFbound, const unsigned int n, const unsigned int ni, const float P_c)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drx, dry, drm, rmada, _1d_drm, _1d_a, c, fsumx, fsumy;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsumx = 0;
		fsumy = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			drx = Ir0[k] + U[j] - U[i];
			dry = Ir0[k + ni] + U[j + n] - U[i + n];
			_1d_a = Ir0[k + 2 * ni];
			c = P_c;

#ifdef pre_nonlinearC
			drm = __fsqrt_rn(drx * drx + dry * dry);
			_1d_drm = __frcp_rn(drm);
			rmada = drm * _1d_a - 1.0f;
			c = dd_nonlinearC(rmada, P_c);
#endif
#ifndef pre_nonlinearC
			_1d_drm = __frsqrt_rn(drx * drx + dry * dry);
#endif // !pre_nonlinearC		

			fsumx += c * drx * (_1d_a - _1d_drm);
			fsumy += c * dry * (_1d_a - _1d_drm);

#ifdef pre_OneCellEdistribution
			drm = __fsqrt_rn(drx * drx + dry * dry);
			rmada = drm - __frcp_rn(_1d_a);
			EFbound[k] = c * _1d_a * rmada * rmada;
#endif // pre_OneCellEdistribution
#ifdef pre_OneCellFdistribution
			drm = __fsqrt_rn(drx * drx + dry * dry);			
			EFbound[k] = c * drm * (_1d_a - _1d_drm);
#endif // pre_OneCellFdistribution
			//if(fsumx* fsumx+ fsumy* fsumy>1e-5)
			//if(idx==4822)
			//	printf("F %i %i %i | %e %e %e | %e %e | %e %e\n", idx, i, j, fsumx, fsumy, _1d_a, U[i], U[i+n], U[j], U[j + n]);
			//printf("FF %e %e %e \n", drx, (1.0 - P_a * _1d_drm), fx[1]);
			//printf("F %u %u | %e %e %e | %e %e | %e %e\n",  j1, idx, fx[1], fy[1], drm, U[j1], U[idx], U[j1 + n], U[idx + n]);

		}
		//if (fsumx * fsumx + fsumy * fsumy > 1e-5)

		//if(idx==4818)
		//	printf("F %u | %e %e %e\n", idx, fsumx, fsumy, fsumx * fsumx + fsumy * fsumy);

		F[idx] = fsumx;
		F[idx + n] = fsumy;
		idx += blockDim.x * gridDim.x;
	}
}