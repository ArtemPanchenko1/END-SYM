#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math_functions.h>
#include "md.h"
#include "pcuda_helper.h"
#include "md_math_constants.h"
#include "md_phys_constants.h"
#include <thrust/count.h>
#include <thrust/device_allocator.h>
#include <thrust/device_ptr.h>
#include <time.h>

//#include <cudpp.h>
//#include <cudpp_plan.h>

__global__ void d_calculateVIscosForces(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, const float* __restrict__ V, float* __restrict__ F, const unsigned int n, const unsigned int ni, const float Po_mu, const float Po_1d_mu, const float Po_rfiber, const float Po_roliquid)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drm, _1d_drm, re;
	float2 dr, t, v, vv, ff, fsum;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsum.x = 0;
		fsum.y = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			dr.x = Ir0[k] + U[j] - U[i];
			dr.y = Ir0[k + ni] + U[j + n] - U[i + n];
			drm = __fsqrt_rn(dr.x * dr.x + dr.y * dr.y);
			_1d_drm = __frcp_rn(drm); //__frsqrt_rn(drx * drx + dry * dry);
			t.x = dr.x * _1d_drm;
			t.y = dr.y * _1d_drm;
			//n.x = t.y; n.y = -t.x;
			v.x = 0.5f * (V[j] + V[i]);
			v.y = 0.5f * (V[j + n] + V[i + n]);
			vv.x = v.x * t.x + v.y * t.y;
			vv.y = v.x * t.y - v.y * t.x;//__fsqrt_rn(v.x * v.x + v.y * v.y - vv.x * vv.x);

			ff.x = 4.0f * Po_rfiber * __fsqrt_rn(MCf_pi * Po_mu * Po_roliquid * drm * fabsf(vv.x)) * vv.x;
			re = fabsf(vv.y) * Po_rfiber * Po_roliquid * Po_1d_mu;
			if (vv.y > 35) printf("ErrorVis %i %i %i | %e %e | %e %e | %e %e \n", idx, i, j, V[i], V[i + n], vv.x, vv.y, re, (2.0f * __logf(1.7811f * 0.25f * re)));
			ff.y = 8.0f * MCf_pi * Po_mu * drm * vv.y * __frcp_rn(1.0f - 2.0f * logf(1.7811f * 0.25f * re));

			fsum.x -= t.x * ff.x + t.y * ff.y;
			fsum.y -= t.y * ff.x - t.x * ff.y;
		}
		F[idx] += 0.5f * fsum.x;
		F[idx + n] += 0.5f * fsum.y;
		idx += blockDim.x * gridDim.x;
	}	
}

__global__ void d_calculateVIscosForces2(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, const float* __restrict__ V, float* __restrict__ F, const unsigned int n, const unsigned int ni, const float Po_mu, const float Po_1d_mu, const float Po_rfiber, const float Po_roliquid, const float Po_1d_hfreefiber)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drm, _1d_drm, re;
	float2 dr, t, v, vv, ff, fsum;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsum.x = 0;
		fsum.y = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			dr.x = Ir0[k] + U[j] - U[i];
			dr.y = Ir0[k + ni] + U[j + n] - U[i + n];
			drm = __fsqrt_rn(dr.x * dr.x + dr.y * dr.y);
			_1d_drm = __frcp_rn(drm); //__frsqrt_rn(drx * drx + dry * dry);
			t.x = dr.x * _1d_drm;
			t.y = dr.y * _1d_drm;
			//n.x = t.y; n.y = -t.x;
			v.x = 0.5f * (V[j] + V[i]);
			v.y = 0.5f * (V[j + n] + V[i + n]);
			vv.x = v.x * t.x + v.y * t.y;
			vv.y = v.x * t.y - v.y * t.x;//__fsqrt_rn(v.x * v.x + v.y * v.y - vv.x * vv.x);

			//ff.x = 4.0f * Po_rfiber * __fsqrt_rn(MCf_pi * Po_mu * Po_roliquid * drm * fabsf(vv.x)) * vv.x;
			ff.x = MCf_pi * Po_rfiber * drm * Po_1d_hfreefiber * Po_mu * vv.x;
			re = fabsf(vv.y) * Po_rfiber * Po_roliquid * Po_1d_mu;
			//if (vv.y > 35) printf("ErrorVis %i %i %i | %e %e | %e %e | %e %e \n", idx, i, j, V[i], V[i + n], vv.x, vv.y, re, (2.0f * __logf(1.7811f * 0.25f * re)));
			ff.y = 8.0f * MCf_pi * Po_mu * drm * vv.y * __frcp_rn(1.0f - 2.0f * logf(1.7811f * 0.25f * re));
#ifdef pre_CylinderDragFluidResistance
			ff.y += 1.1f * Po_roliquid * Po_rfiber * drm * fabsf(vv.y) * vv.y;
#endif // pre_CylinderDragFluidResistance

			fsum.x -= t.x * ff.x + t.y * ff.y;
			fsum.y -= t.y * ff.x - t.x * ff.y;
		}
		F[idx] += 0.5f * fsum.x;
		F[idx + n] += 0.5f * fsum.y;
		idx += blockDim.x * gridDim.x;
	}
}

__global__ void d_calculateVIscosForces3(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, const float* __restrict__ V, float* __restrict__ F, const unsigned int n, const unsigned int ni, const float Po_mu, const float Po_1d_mu, const float Po_rfiber, const float Po_roliquid, const float Po_1d_hfreefiber)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drm, _1d_drm, re, c, m_1d_dv, vabs;
	float2 dr, t, vi, vj, ff, fsum;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsum.x = 0;
		fsum.y = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			dr.x = Ir0[k] + U[j] - U[i];
			dr.y = Ir0[k + ni] + U[j + n] - U[i + n];
			drm = __fsqrt_rn(dr.x * dr.x + dr.y * dr.y);
			_1d_drm = __frcp_rn(drm); //__frsqrt_rn(drx * drx + dry * dry);
			t.x = dr.x * _1d_drm;
			t.y = dr.y * _1d_drm;

			vi.x = V[i] * t.x + V[i + n] * t.y;
			vi.y = V[i] * t.y - V[i + n] * t.x;
			vj.x = V[j] * t.x + V[j + n] * t.y;
			vj.y = V[j] * t.y - V[j + n] * t.x;
			c = 0.5f * MCf_pi * Po_rfiber * drm * Po_1d_hfreefiber * Po_mu;
			vabs = fabsf(vi.x) + fabsf(vj.x);			
			if (fabsf(vi.x - vj.x) > 1e-9)
				m_1d_dv = vi.x * __frcp_rn(vi.x - vj.x);
			else 
				m_1d_dv = 0;
			if (m_1d_dv > 0 && m_1d_dv < 1)
			{
				ff.x = c * vi.x * m_1d_dv;
			}
			else if (vabs > 1e-9)
			{
				ff.x = c * (vi.x + vj.x) * fabsf(vi.x) * __frcp_rn(vabs);
			}
			else
			{
				ff.x = 0.5f * c * (vi.x + vj.x);
			}
			re = 0.5f * vabs * Po_rfiber * Po_roliquid * Po_1d_mu;
			ff.y = 8.0f * MCf_pi * MCf_1d6 * Po_mu * drm * (2.0f * vi.y + vj.y) * __frcp_rn(1.0f - 2.0f * logf(1.7811f * 0.25f * re));

			fsum.x -= t.x * ff.x + t.y * ff.y;
			fsum.y -= t.y * ff.x - t.x * ff.y;
		}
		F[idx] += fsum.x;
		F[idx + n] += fsum.y;
		idx += blockDim.x * gridDim.x;
	}
}

__global__ void d_calculateVIscosForcesShapovalov(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, const float* __restrict__ V, float* __restrict__ F, const unsigned int n, const unsigned int ni, const float Po_mu, const float Po_1d_mu, const float Po_rfiber, const float Po_roliquid, const float Po_CShfreefiber)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drm, _1d_drm, _1d_re, c, m_1d_dv, vabs;
	float2 dr, t, vi, vj, ff, fsum;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsum.x = 0;
		fsum.y = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			dr.x = Ir0[k] + U[j] - U[i];
			dr.y = Ir0[k + ni] + U[j + n] - U[i + n];
			drm = __fsqrt_rn(dr.x * dr.x + dr.y * dr.y);
			_1d_drm = __frcp_rn(drm); //__frsqrt_rn(drx * drx + dry * dry);
			t.x = dr.x * _1d_drm;
			t.y = dr.y * _1d_drm;

			vi.x = V[i] * t.x + V[i + n] * t.y;
			vi.y = V[i] * t.y - V[i + n] * t.x;
			vj.x = V[j] * t.x + V[j + n] * t.y;
			vj.y = V[j] * t.y - V[j + n] * t.x;

			c = MCf_pi * Po_rfiber * drm * Po_mu * Po_CShfreefiber;
			vabs = fabsf(vi.x) + fabsf(vj.x);
			if (fabsf(vi.x - vj.x) > 1e-9)
				m_1d_dv = vi.x * __frcp_rn(vi.x - vj.x);
			else
				m_1d_dv = 0;
			if (m_1d_dv > 0 && m_1d_dv < 1)
			{
				ff.x = c * vi.x * m_1d_dv;
			}
			else if (vabs > 1e-9)
			{
				ff.x = c * (vi.x + vj.x) * fabsf(vi.x) * __frcp_rn(vabs);
			}
			else
			{
				ff.x = 0.5f * c * (vi.x + vj.x);
			}
			_1d_re = __frcp_rn(0.5f * vabs * Po_rfiber * Po_roliquid * Po_1d_mu);
			ff.y = 4.0f * MCf_pi * MCf_1d6 * Po_mu * drm * (2.0f * vi.y + vj.y) * __frcp_rn(logf(7.4f * _1d_re));
			//ff.x = 0;
			//ff.y = 0;
			fsum.x -= t.x * ff.x + t.y * ff.y;
			fsum.y -= t.y * ff.x - t.x * ff.y;
		}
		F[idx] += fsum.x;
		F[idx + n] += fsum.y;
		idx += blockDim.x * gridDim.x;
	}
}

__global__ void d_calculateVIscosForcesLindstrom(const int* __restrict__ In, const float* __restrict__ Ir0, const int* __restrict__ ShIn, const float* __restrict__ U, const float* __restrict__ V, float* __restrict__ F, const unsigned int n, const unsigned int ni, const float Po_mu, const float Po_rfiber, const float Po_1d_rfiber, const float Po_roliquid)
{
	//int vpx, vpy, j1, j2, mj1, mj2;
	int i, j, k, ks, kmax;
	float drm, _1d_drm, a,b,e,L,XA, YA, YC, c1, c2, c3, re, c, m_1d_dv, vabs;
	float2 dr, t, vi, vj, ff, fsum;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("IM %i %i %i %i %i %i %i\n", IM[0], IM[1], IM[2], IM[3], IM[4], IM[5], IM[6]);
	while (idx < n)
	{
		//printf("FI %i %i %i\n", idx, ShIn[idx], ShIn[idx + n]);
		ks = ShIn[idx];
		kmax = ks + ShIn[idx + n];
		fsum.x = 0;
		fsum.y = 0;
		for (k = ks; k < kmax; ++k)
		{
			i = In[k];
			j = In[k + ni];

			dr.x = Ir0[k] + U[j] - U[i];
			dr.y = Ir0[k + ni] + U[j + n] - U[i + n];
			drm = __fsqrt_rn(dr.x * dr.x + dr.y * dr.y);
			_1d_drm = __frcp_rn(drm); //__frsqrt_rn(drx * drx + dry * dry);
			t.x = dr.x * _1d_drm;
			t.y = dr.y * _1d_drm;

			vi.x = V[i] * t.x + V[i + n] * t.y;
			vi.y = V[i] * t.y - V[i + n] * t.x;
			vj.x = V[j] * t.x + V[j + n] * t.y;
			vj.y = V[j] * t.y - V[j + n] * t.x;

			a = 0.5f * drm;
			b = 0.806451612903225f * Po_rfiber * __fsqrt_rn(logf(0.5f * drm * Po_1d_rfiber));
			e = __fsqrt_rn(1.0f - 4.0f * b * b * _1d_drm * _1d_drm);
			L = logf((1.0f + e) * __frcp_rn(1.0f - e));
			c1 = 4.0f * MCf_pi * MCf_1d3 * e * e * e * drm * Po_mu;
			c2 = __frcp_rn(-2.0f * e + (1.0f + e * e) * L);
			c3 = __frcp_rn(2.0f * e + (3.0f * e * e - 1.0f) * L);
			XA = 6.0f * c1 * c2;
			YA = 12.0f * c1 * c3;
			YC = c1 * (2.0f - e * e) * c2;

			vabs = fabsf(vi.x) + fabsf(vj.x);
			if (vabs > 1e-9)
			{
				ff.x = 0.5f * XA * (vi.x + vj.x) * fabsf(vi.x) * __frcp_rn(vabs);
			}
			else
			{
				ff.x = 0.25f * XA * (vi.x + vj.x);
			}
			//ff.x = XA * 0.25f * (vi.x + vj.x);
			ff.y = YA * (vi.y + vj.y) * 0.25f - YC * (vj.y - vi.y);
			//if(idx==568)printf("IM %f %f | %f %f %f %f %f %f %f\n", drm, Po_rfiber, a, b, e, L, XA, YA, YC);
			//c = MCf_pi * 2.0f * Po_rfiber * drm * Po_mu * Po_CShfreefiber;
			//c = 3.0f * XA;
			//re = (fabsf(vi.y) + fabsf(vj.y)) * Po_rfiber * Po_roliquid * Po_1d_mu;
			//ff.y = 4.0f * MCf_pi * Po_mu * drm * (2.0f * vi.y + vj.y) * __frcp_rn(logf(7.4f * re));
			//ff.y = 0;
			fsum.x -= t.x * ff.x + t.y * ff.y;
			fsum.y -= t.y * ff.x - t.x * ff.y;
		}
		F[idx] += fsum.x;
		F[idx + n] += fsum.y;
		idx += blockDim.x * gridDim.x;
	}
}